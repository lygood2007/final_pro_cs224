
#ifndef TEST_CU
#define TEST_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

extern "C"
void testVector();

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void testVector() {
        int N = 5;
        size_t size = N * sizeof(float);

        // Allocate input vectors h_A and h_B in host memory
        float* h_A = (float*)malloc(size);
        float* h_B = (float*)malloc(size);
        float* h_C = (float*)malloc(size);

        // Initialize input vectors.

        // Allocate vectors in device memory
        void* amp;
        float* d_A;
        hipMalloc(&d_A, size);
        float* d_B;
        hipMalloc(&d_B, size);
        float* d_C;
        hipMalloc(&d_C, size);
        for( int i = 0; i < N; i++ )
        {
            h_A[i] = 2;
            h_B[i] = 3;
        }

        // Copy vectors from host memory to device memory
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        // Invoke kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (N+threadsPerBlock-1)/threadsPerBlock;
        VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

        // Copy result from device memory to host memory
        // h_C contains the result in host memory
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

     //   assert( d_A[0]+d_B[0] == d_C[0]&&"Cuda is not running or has problems" );

        fflush(stdin);
        fflush(stdout);
        for( int i = 0; i < N; i++ )
        {
//            printf("%f", h_C[i] );
        }
        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
}

#endif
