
#ifndef TEST_CU
#define TEST_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

extern "C"
{
    void testVector();
    bool findSupportDevice();
}
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void testVector() {
        int N = 5;
        size_t size = N * sizeof(float);

        // Allocate input vectors h_A and h_B in host memory
        float* h_A = (float*)malloc(size);
        float* h_B = (float*)malloc(size);
        float* h_C = (float*)malloc(size);

        // Initialize input vectors.

        // Allocate vectors in device memory
        void* amp;
        float* d_A;
        hipMalloc(&d_A, size);
        float* d_B;
        hipMalloc(&d_B, size);
        float* d_C;
        hipMalloc(&d_C, size);
        for( int i = 0; i < N; i++ )
        {
            h_A[i] = 2;
            h_B[i] = 3;
        }

        // Copy vectors from host memory to device memory
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        // Invoke kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (N+threadsPerBlock-1)/threadsPerBlock;
        VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

        // Copy result from device memory to host memory
        // h_C contains the result in host memory
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

     //   assert( d_A[0]+d_B[0] == d_C[0]&&"Cuda is not running or has problems" );

        fflush(stdin);
        fflush(stdout);
        for( int i = 0; i < N; i++ )
        {
//            printf("%f", h_C[i] );
        }
        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
}

/**
 * @brief findSupportGPU Find supported CUDA device counts
 * @return True if device count is not zero
 */
bool findSupportDevice()
{
       int deviceCount = 0;

       hipError_t error_id = hipGetDeviceCount(&deviceCount);

       if (error_id != hipSuccess)
       {
           printf("hipGetDeviceCount returned error code: %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
           printf("> FAILED %s sample finished, exiting...\n" );
           exit(EXIT_FAILURE);
       }
       if (deviceCount == 0)
       {
           printf("> There are no device(s) supporting CUDA\n");
           return false;
       }
       else
       {
           printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
       }
}

#endif
