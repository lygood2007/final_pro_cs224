#include "hip/hip_runtime.h"
/** fluid_compute.cu
 ** Brief: Deal with all the computation here
 ** Project: large-scale fluids
 ** Date: 04/10/2013
 ** Member: Scott, Hobarts, Yan Li
 **/

#ifndef FLUID_COMPUTE_CU
#define FLUID_COMPUTE_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include "fluid_global.h"
// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define EPS 0.001
#define DEC 0.1
extern "C"
{
void initGridGPU( const int hostGridSize, const int hostGridPaintSize, const float hostdx, const float halfdm, const float* hostTerrainMap );
void copybackGPU(FieldType type, float* hostMap  );
void destroyGPUmem();
void addDropGPU(const int posX, const int posZ, const int radius, const float h );
void addSideWaveGPU(const int sideIndex, const int sideLength, const float h );
void advectGPU(const float dt);
void updateFluidGPU( const float dt );
bool findSupportDevice();

void initParticlesGPU(const float minHeight, const int numSplashParticles, const int numSprayParticles, const int numFoamParticles);
void updateParticlesGPU( const float minHeight, const float dt, const float halfDomain, const float mdxInv, const float accX, const float accY, const float accZ );
void intersectParticlesGPU( const float minHeight, const float halfDomain, const float mdx, const float mdxInv,
                            const float splashVeff, const float splashHeightChange, const float sprayVeff, const float sprayHeightChange );
void inputParticlesGPU(const float *particlePositions, const float *particleVelocities );
void inputSprayParticlesGPU( const float *particlePositions, const float *particleVelocities );
void inputFoamParticlesGPU( const float *particlePositions, const float *ttlArray );

void checkBreakingWavesGPU( const float condition1, const float condition2, const float condition3,
                            const float mdxInv, const float dt );
void inputDepthGPU( const float* newDepthField );

void clampFieldsGPU( const float velocityClamp );

void initDampeningFieldsGPU( const int sizeDampeningRegion, const float quadraticA, const float quadraticB, const float quadraticC );
void dampenWavesGPU( const float hRest, const float dt, const float dxInv, const float lambdaUpdate, const float lambdaDecay );
}

__host__ __device__ inline float cudaMax( float a, float b )
{
    return (a>b)?(a):(b);
}

__host__ __device__ inline float cudaMin( float a, float b )
{
    return (a<b)?(a):(b);
}


// The 2D vector structure for GPU computing
 struct vec2
{
    union
    {
        struct {float x,y; };
        float xy[2];
    };
};

 struct vec3
{
    union
    {
        struct {float x,y,z; };
        float xyz[3];
    };
};

const int blockSizeX = 4;
const int blockSizeY = 4;

vec3* devicePaintMap; // Paint map, we copy back this buffer for drawing. It stores the position of vertices
vec3* devicePaintNormalMap; // Normal map for GPU
float* deviceTerrainMap; // Terrain map for GPU
float* deviceHeightMap; // Height map for GPU
float* deviceDepthMap; // Depth map for GPU
float* devicePrevDepthMap; // Depth map of previous timestep for GPU
float* deviceVelocityUMap; // VelocityU map for GPU
float* deviceVelocityWMap; // VelocityW map for GPU

float* deviceNextDepthMap; // Temp buffer for storing next depth map
float* deviceNextVelocityUMap; // Temp buffer for storing next velocity U map
float* deviceNextVelocityWMap; // Temp buffer for storing next velocity W map

//particle data structures
vec3* deviceParticlePositionsArray; // particle positions array (splash)
vec3* deviceParticleVelocitiesArray; // particle velocities array (splash)
vec3* deviceSprayPositionsArray; // spray positions array
vec3* deviceSprayVelocitiesArray; // spray velocities array
vec3* deviceFoamPositionsArray; // foam positions array
float* deviceFoamTTLArray; // foam time-to-live array
float* deviceSplashToFoamArray; // array to alert the user to turn splash into foam
float* deviceBreakingWavesMap; // map of grid cells that are breaking, number of particles to instantiate

int deviceNumSplashParticles; // number of splash particles
int deviceNumSprayParticles; // number of spray particles
int deviceNumFoamParticles; // number of foam particles

//dampening waves data structures
float* deviceSigmaMap;
float* deviceGammaMap;
float* devicePhiMap;
float* devicePsiMap;
int deviceDampeningRegion; // size of dampening region

/**
 * pitches for the maps above
 */
// Error
hipError_t error;

// The grid size for heightmap, depthmap, terrainmap
int gridSize;
// The grid size for paint
int gridPaintSize;
// The width for velocity u
int uwidth;
// The height for veloctiy u
int uheight;
// The width for velocity w
int wwidth;
// The height for veloctiy w
int wheight;


// dx
float mapdx;
// dxInv
float mapdxInv;
// halfDomain
float halfDomain;

void checkInitializedDeviceField( float* device, int width, int height )
{
    float* host = (float*)malloc(width*height*sizeof(float));
    hipMemcpy( host, device, width*height*sizeof(float),hipMemcpyDeviceToHost);

    for( int i = 0; i < width*height; i++ )
    {
        if( host[i] != 0.f )
        {
            assert(0);
        }
    }
    free(host);
}

void  checkCudaError( hipError_t error )
{
    if( error != hipSuccess )
    {
        //cout <<"CUDA error code: "<<hipGetErrorString(error);
        printf( "CUDA error code: %s\n",hipGetErrorString(error) );
    }
}
/**
 * Check if the pointer is null, if it's null, exit the program
 */
template <class T>
void check1DNotNull(T* array )
{
    if( array == NULL )
    {
        printf("Wrong pointer\n");
    }
}

/**
 * Initialize a vec2
 */
__host__ __device__ inline vec2 initVec2( float x, float y )
{
    vec2 result;
    result.x = x;result.y = y;
    return result;
}

/**
 *  Initialize a vec3
 */
__host__ __device__ inline vec3 initVec3( float x, float y, float z )
{
    vec3 result;
    result.x = x;result.y = y; result.z = z;
    return result;
}

// Review passed
/**
* Compute the cross product of two vectors
*/
__host__ __device__ inline vec3 cross( const vec3 v1, const vec3 v2 )
{
    vec3 result = initVec3(v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z, v1.x * v2.y - v1.y * v2.x);
    return result;
}

// Review passed
/**
 * Compute the normalized vector
 */
__host__ __device__ inline vec3 normalize( const vec3 v )
{
    float d = sqrt(v.x*v.x + v.y*v.y + v.z*v.z );
    return initVec3( v.x/d,v.y/d, v.z/d );
}

// Review passed
/**
 * Functions for getting data from 2D array in GPU
 */
__host__ __device__ inline float map2Dread( const float* map, int i, int j, int width )
{
    return map[i*width+ j];
}
// Review passed
/**
 * Functions for writting data from 2D array in GPU
 */
__host__ __device__ inline void map2Dwrite( float* map, int i, int j, float value, int width )
{
    map[i*width + j] = value;
}

// Review passed
/**
 *  Initialize the depth
 */
__global__ void initDepthCUDA( float* depthMap, const float* terrainMap, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;

    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        float depth = cudaMax(0.f,defaultHeight - map2Dread( terrainMap, i,j, width ));
       // float depth = 5;
        map2Dwrite( depthMap,i,j,depth,width );
    }
}

// Review passed
/**
 * Update the height field by plusing the depth and terrain
 */
__global__ void updateHeightCUDA( float*heightMap, const float* depthMap, const float* terrainMap,
                              const int width, const int height )
{
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        float depth = map2Dread( depthMap,i,j,width );
        float terrainHeight = map2Dread( terrainMap, i,j,width );
        float h = depth + terrainHeight;
        map2Dwrite( heightMap, i,j, h ,width);
    }
}

// Review passed
/**
 * Add drop to specified rectangular region
 */
__global__ void addDropCUDA( float* depthMap, const int posX, const int posZ, const int radius,
                             const float h, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

//    if( i>= cudaMax(posZ-radius,0) && i < cudaMin(posZ+radius+1,height)
//            && j >= cudaMax(posX-radius,0)&&j < cudaMin(posX+radius+1,width)
//            )
//    {
//        float newH = cudaMin(map2Dread(depthMap,i,j,width)+h,maxHeight );
//        map2Dwrite( depthMap, i,j, newH, width );
//    }

    float iDistance = (float) (i - posZ);
    float jDistance = (float) (j - posX);
    float distance = sqrt((iDistance * iDistance) + (jDistance * jDistance));
    if(distance <= (float) radius){
        float newH = cudaMin(map2Dread(depthMap,i,j,width)+h,maxHeight );
        map2Dwrite( depthMap, i,j, newH, width );
    }
}

/**
 * Add drop to specified side
 */
__global__ void addSideWaveCUDA( float* depthMap, const int sideLength, const int sideIndex,
                             const float h, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    if(sideIndex == 1){
        //side index 1: min i
        if( i >= 0 && i < sideLength && j >= 0 && j < width )
        {
            float newH = cudaMin(map2Dread(depthMap,i,j,width)+h,maxHeight );
            map2Dwrite( depthMap, i,j, newH, width );
        }
    } else if(sideIndex == 2){
        //side index 2: max i
        if( i >= height - sideLength && i < height && j >= 0 && j < width )
        {
            float newH = cudaMin(map2Dread(depthMap,i,j,width)+h,maxHeight );
            map2Dwrite( depthMap, i,j, newH, width );
        }
    } else if(sideIndex == 3){
        //side index 3: min j
        if( i >= 0 && i < height && j >= 0 && j < sideLength )
        {
            float newH = cudaMin(map2Dread(depthMap,i,j,width)+h,maxHeight );
            map2Dwrite( depthMap, i,j, newH, width );
        }
    } else if(sideIndex == 4){
        //side index 4: max j
        if( i >= 0 && i < height && j >= width - sideLength && j < width )
        {
            float newH = cudaMin(map2Dread(depthMap,i,j,width)+h,maxHeight );
            map2Dwrite( depthMap, i,j, newH, width );
        }
    }
}

// Review passed
/**
 * bilinear interpolation
 */
 __host__ __device__ float bilinearIerp( const float* vec, float x, float z, const int width, const int height )
 {
     if( x < 0 )
         x = 0.f;
     if( z < 0 )
         z = 0.f;
     if( x > width - 1 )
         x = width - 1;
     if( z > height - 1 )
         z = height -1;

     const int X = (int)x;
     const int Y = (int)z;
     const float s1 = x - X;
     const float s0 = 1.f - s1;
     const float t1 = z - Y;
     const float t0 = 1.f-t1;
     float e1, e2, e3,e4;
     e1 = e2 = e3 = e4 = 0;
     //e1 = vec[Y*width+X];
     e1 = map2Dread(vec,Y,X,width);
     if( Y+1 <= height- 1 )
     {
      //   e2 = vec[(Y+1)*width + X];
         e2 = map2Dread( vec, Y+1,X,width);
     }
     if( X +1 <= width -1 )
     {
        // e3 = vec[Y*width + X+1];
         e3 = map2Dread( vec, Y, X+1, width );
     }
     if( Y+1 <= height - 1 && X + 1 <= width - 1)
     {
      //   e4 = vec[(Y+1)*width + X+1];
         e4 = map2Dread( vec, Y+1,X+1,width );
     }

     float result = s0*(t0*e1 + t1*e2 )+
             s1*(t0*e3  + t1*e4 );

     return  result;
 }

 // Review passed
 /**
  * Advection: depth
  **/
 __global__ void advectDepthCUDA( const float* depthMap, float* nextDepthMap, const float* velUMap, const float* velWMap,
                              const int width, const int height, const float dt, const float dxInv)
{
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height -1 && j >= 1 && j < width - 1 )
     {
         // Be careful about the width of velU
         float uw = width + 1;
         float u = 0.5*(map2Dread(velUMap,i,j,uw) + map2Dread( velUMap,i,j+1,uw) );
         float w = 0.5*(map2Dread(velWMap,i,j,width) + map2Dread( velWMap,i+1,j,width) );

         float curPosX = (float)j;
         float curPosY = (float)i;
         float prev_x = curPosX - u*dt*dxInv;
         float prev_z = curPosY - w*dt*dxInv;
         map2Dwrite(nextDepthMap, i,j, bilinearIerp( depthMap,prev_x,prev_z,width,height ), width);
     }
 }

 // Review passed
 /**
  * Advection: velocity U
  */
 __global__ void advectVelUCUDA( const float* velUMap, float* nextVelUMap, const float* velWMap,
                            const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height - 1&&j >= 1 && j < width - 1)
     {
         float ww = width - 1;
        float u = map2Dread( velUMap, i,j,width );
        float w = 0.25*(map2Dread(velWMap,i,j, ww ) + map2Dread( velWMap, i,j-1,ww) + map2Dread( velWMap,i+1,j-1,ww) + map2Dread( velWMap, i+1,j,ww ) );
        float curPosX = (float)j;
        float curPosY = (float)i;
        float prev_x = curPosX - u*dt*dxInv;
        float prev_z = curPosY - w*dt*dxInv;
        map2Dwrite(nextVelUMap, i,j, bilinearIerp(  velUMap,prev_x,prev_z,width,height ), width);
     }
 }

 // Review passed
 /**
  *  Advection: velocity W
  */
 __global__ void advectVelWCUDA( const float* velWMap, float* nextVelWMap, const float* velUMap,
                            const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height - 1&&j >= 1 && j < width - 1)
     {
        float uw = width + 1;
        float u = 0.25*(map2Dread( velUMap,i,j,uw) + map2Dread(velUMap,i,j+1,uw) + map2Dread(velUMap,i-1,j+1,uw) + map2Dread(velUMap, i-1,j,uw) );
        float w = map2Dread( velWMap,i,j,width );
        float curPosX = (float)j;
        float curPosY = (float)i;
        float prev_x = curPosX - u*dt*dxInv;
        float prev_z = curPosY - w*dt*dxInv;
        map2Dwrite(nextVelWMap,i,j,bilinearIerp(  velWMap,prev_x,prev_z,width,height ), width);
     }
 }

 // Review passed
 /**
  * Update the depth field
  **/
 __global__ void updateDepthCUDA(float* depthMap, const float* velUMap, const float* velWMap,
                                 const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height-1 && j >= 1 && j < width-1)
     {
            float decay = 1.f;
            float dep = map2Dread(depthMap,i,j,width);
            float dh = -decay*dep*dxInv*( (map2Dread(velUMap,i,j+1,width+1) - map2Dread(velUMap,i,j,width+1))
                                          + (map2Dread(velWMap,i+1,j,width) - map2Dread(velWMap,i,j,width)) );
            float nextDepth = dh*dt+dep;
            if( nextDepth < EPS )
            {
                 map2Dwrite(depthMap, i, j, 0.f, width );
            }
            else
            {
                map2Dwrite(depthMap, i, j, dh*dt+dep, width );
            }
     }
 }

 // Review passed
 /**
  * Update the velocity U field
  */
 __global__ void updateVelUCUDA( float* velUMap, const float* heightMap, const float* depthMap,
                                 const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >=1 && i < height - 1 && j >= 2 && j < width - 2 )
     {
         // The width of heightmap is 1 smaller than the width of velocity U
        float h1 = map2Dread( heightMap, i,j, width - 1);
        float h2 = map2Dread(  heightMap, i,j-1, width-1 );
        float d1 = map2Dread(depthMap,i,j,width-1);
        float d2 = map2Dread(depthMap,i,j-1,width-1);

        // Read the origin value from velUMap
        float vel = map2Dread( velUMap, i,j, width );

        if( d1 < EPS || d2 < EPS )
        {
            float vel1 = map2Dread( velUMap,i,j-1,width);
            float vel2 = map2Dread( velUMap,i,j+1,width);
            float vel3 = map2Dread( velUMap, i,j, width );
            map2Dwrite( velUMap,i,j,0.33*(vel1+vel2+vel3),width);
            return;

        }
        float dv = GRAVITY*dt*dxInv*(h1-h2);

        // Add
        map2Dwrite( velUMap, i,j,vel+dv, width);
     }
     else
     {
         // for the bounday, we set the velocity to zero, using the neuman boundary condition
        map2Dwrite( velUMap, i,j,0.f, width);
     }
 }

 // Review passed
 /**
  * Update the velocity W field
  */
 __global__ void updateVelWCUDA( float* velWMap, const float* heightMap, const float* depthMap,
                                 const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 2 && i < height - 2 && j >= 1 && j < width - 1 )
     {
         float h1 = map2Dread( heightMap, i,j, width );
         float h2 = map2Dread(  heightMap, i-1,j, width );

         float d1 = map2Dread(depthMap,i,j,width);
         float d2 = map2Dread(depthMap,i-1,j,width);

         float vel = map2Dread( velWMap, i,j, width );

         if( d1 < 0.0001 || d2 < 0.0001 )
         {
             float vel1 = map2Dread( velWMap,i-1,j,width);          
             float vel2 = map2Dread( velWMap,i+1,j,width);
             float vel3 = map2Dread( velWMap, i, j, width );
             map2Dwrite( velWMap,i,j,0.33*(vel1+vel2+vel3),width);
             return;

         }
         float dv = GRAVITY*dt*dxInv*(h1-h2);
         map2Dwrite( velWMap, i,j,vel+dv, width );
     }
     else
     {
         // for the bounday, we set the velocity to zero, using the neuman boundary condition
        map2Dwrite( velWMap, i,j,0.f, width);
     }
 }

 // Review passed
 /**
  * Apply the boundary condition
  **/
 __global__ void applyBoundaryCUDA( float* depthMap, const float* heightMap, const float* terrainMap,
                                    const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
    float value;
    // Fix the boundary
    /*if( i == 0 || i == height-1|| j == 0 ||  j == width-1 )
    {
        map2Dwrite( depthMap, i,j, 0, width );
    }*/
     if( j == 0 && i !=  0 && i != height-1)
     {
         value = max(0.f, map2Dread( heightMap, i,1,width) - map2Dread( terrainMap, i,j,width ) );
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if( j== width - 1&& i !=0 && i != height - 1 )
     {
        value = max( 0.f, map2Dread( heightMap, i, width - 2, width ) - map2Dread( terrainMap, i, j,width) );
        map2Dwrite( depthMap, i, j, value, width );
        return;
     }

     if( i == 0&& j != 0 && j != width -1 )
     {
         value = max(0.f, map2Dread( heightMap, 1, j, width) - map2Dread( terrainMap, i, j,width ));
         map2Dwrite( depthMap, i, j, value, width );
         return;
     }
     else if( i == height - 1&& j != width - 1 && j != 0)
     {
         value = max(0.f, map2Dread( heightMap, height - 2, j, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i, j, value, width );
         return;
     }

     // Deal with the four courner, is there a way to simplify this? This function is too long!
     if( i== 0 && j == 0 )
     {
         value = max(0.f, map2Dread( heightMap, 1, 1, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if( i==0 && j == width - 1 )
     {
         value = max(0.f, map2Dread( heightMap, 1, width-2, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if ( i == height - 1 && j == 0 )
     {
         value = max(0.f, map2Dread( heightMap, height - 2, 1, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if( i == height - 1 && j == width - 1 )
     {
         value = max(0.f, map2Dread( heightMap, height - 2, width-2, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
 }

 // Review passed
 /**
  * A general function to initialize the field to be all zero( Cannot use memset because we are using float )
  **/
 __global__ void initFieldCUDA( float* deviceMap, int width, int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
         map2Dwrite( deviceMap, i, j, 0.f, width );
     }
 }

 // Review passed
/**
 * Initialize the normal field
 **/
 __global__ void initPaintNormalCUDA( vec3* paintNormalMap, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
         const int index = i*width + j;
         paintNormalMap[index].x = 0;
         paintNormalMap[index].y = 1;
         paintNormalMap[index].z = 0;
     }
 }

 /**
  * Update the paint field
  */
 __global__ void updatePaintCUDA( vec3* paintMap, const float* heightMap,  const float* depthMap, const float halfdm,
                                  const float dx, const int gSize )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < gSize&&j >= 0 && j < gSize )
     {
         const int index = i*gSize+ j;
         paintMap[index].x = -halfdm + j*dx;     
         paintMap[index].y = heightMap[index];
         if( depthMap[index] < EPS )
             paintMap[index].y -= DEC;
         paintMap[index].z = -halfdm + i*dx;
     }
 }

 // Review passed
 /**
  * Update the paint field with boundary included
  */
 __global__ void updatePaintBoundCUDA( vec3* paintMap, const float* heightMap,  const float* depthMap,
                                       const float halfdm, const float dx, const int gSize )
 {
     // Height must be same as width
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < gSize&&j >= 0 && j < gSize )
     {
         // current index
         const int curInd = i*gSize + j;
         // If it is not in boundary
         int ii;
         int jj;
         if( i >= 1 && j >= 1 && i <= gSize - 2 && j <= gSize - 2)
         {
             ii = i-1;
             jj = j-1;
             paintMap[curInd].x = -halfdm + jj*dx;
             paintMap[curInd].y = map2Dread( heightMap, ii, jj , gSize-2 );
             if( map2Dread(depthMap,ii,jj,gSize-2) < EPS )
             {
                 // We decrease the surface for painting
                 paintMap[curInd].y -= DEC;
             }
             paintMap[curInd].z = -halfdm + ii*dx;
         }
         else // Boundary condition
         {
            if(i == 0 && j == 0 ) // Left top corner
            {
                ii = 0;
                jj = 0;
            }
            else if( i == gSize - 1 && j == gSize - 1 ) // Right bottom corner
            {
                ii = i-2;
                jj = j-2;
            }
            else if( i == 0 && j == gSize - 1 ) // Right top corner
            {
                ii = 0;
                jj = j-2;
            }
            else if( i == gSize - 1 && j== 0 ) // Left bottom corner
            {
                ii = i-2;
                jj = 0;
            }
            else
            {
                if( i == 0)
                {
                    ii = 0;
                    jj = j-1;
                }
                else if( i == gSize - 1 )
                {
                    ii = i-2;
                    jj = j -1;

                }
                else if( j == 0 )
                {
                    ii = i-1;
                    jj = 0;
                }
                else if( j== gSize - 1 )
                {
                    ii = i-1;
                    jj = j-2;
                }
            }

            paintMap[curInd].x = -halfdm + jj*dx;
            paintMap[curInd].y = map2Dread(heightMap,ii,jj,gSize-2 ) - map2Dread( depthMap,ii,jj,gSize-2);
            paintMap[curInd].z = -halfdm + ii*dx;
         }
     }
 }

// Review passed
 /**
  * compute the normals
  **/
 __global__ void computePaintNormalCUDA( vec3* paintNormalMap, const vec3* paintMap, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
            int numNeighbours = 0;
            const int currInd = i*width + j;
            vec3 offset[8];
            vec2 coords[8];
            vec2 neighbours[8];

            coords[0] = initVec2(i,     j - 1);
            coords[1] = initVec2(i + 1, j - 1);
            coords[2] = initVec2(i + 1, j);
            coords[3] = initVec2(i + 1, j + 1);
            coords[4] = initVec2(i,     j + 1);
            coords[5] = initVec2(i - 1, j + 1);
            coords[6] = initVec2(i - 1, j);
            coords[7] = initVec2(i - 1, j - 1);
            int m;
            for( m = 0; m < 8; m++ )
            {
                if( coords[m].x < 0 || coords[m].y < 0 || coords[m].x > height- 1 || coords[m].y > width- 1 )
                    continue;
                neighbours[numNeighbours] = coords[m];
                numNeighbours++;
            }

            for( m = 0; m < numNeighbours; m++ )
            {
                /*offset[m].x = neighbours[m].y - j;
                offset[m].z = neighbours[m].x - i;
                offset[m].x = map2Dread( )
                offset[m].y = map2Dread(heightMap,neighbours[m].x,neighbours[m].y,width) - map2Dread( heightMap, i,j,width );
                */
                const int ind1 = neighbours[m].x*width + neighbours[m].y;

                offset[m].x = paintMap[ind1].x - paintMap[currInd].x;
                offset[m].y = paintMap[ind1].y - paintMap[currInd].y;
                offset[m].z = paintMap[ind1].z - paintMap[currInd].z;
            }

            vec3 sum = initVec3(0.f,0.f,0.f);
            for( m = 0; m < numNeighbours; m++ )
            {
                vec3 tmp;
                if( m+1 == numNeighbours )
                    tmp = cross( offset[m],offset[0]);
                else
                    tmp = cross( offset[m],offset[m+1]);
                sum.x += tmp.x;
                sum.y += tmp.y;
                sum.z += tmp.z;
            }
            vec3 result = normalize( sum );
           paintNormalMap[currInd].x = result.x; paintNormalMap[currInd].y = result.y; paintNormalMap[currInd].z = result.z;
     }
 }

 // Review not done
 /**
  * Reduce the overshooting phenomenon when the wave enters a shallow region
  */
 __global__ void overshootingReduction( const float* depthMap, float* nextDepthMap, const float* heightMap,
                                        const float dx, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0 && i < height && j >= 0 && j < width )
     {
         float hij = map2Dread( depthMap, i,j,width );
         // Exclude the border
         if( i >= 1 && i < height-1 && j >= 1 && j < width-1 )
         {
             // 2.2 section
             const float alpha = 0.3;
             // n(i,j)
             float n = map2Dread( heightMap, i,j,width );
             // n(i-1,j)
             float n2 = map2Dread( heightMap, i-1,j,width );
             // n(i+1,j)
             float n3 = map2Dread( heightMap, i+1,j,width );
             // n(i,j-1)
             float n4 = map2Dread( heightMap, i, j-1, width );
             // n(i,j+1)
             float n5 = map2Dread( heightMap, i, j+1, width );
             float value;
             float nextD = hij;
             float lamda = 2*dx;
             if( n - n2 >lamda && n > n3  )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap,i+1,j,width) ) ) - hij );
                 nextD += value;
             }
             if( n - n3 > lamda && n > n2 )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap,i-1,j,width) ) ) - hij );
                 nextD += value;
             }
             if( n - n4 > lamda && n > n5 )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap, i,j+1,width ) ) ) - hij );
                 nextD += value;
             }
             if( n - n5 > lamda && n > n4 )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap, i, j- 1, width) ) ) - hij );
                 nextD += value;
             }
             map2Dwrite( nextDepthMap, i,j, nextD,width );
         }
         else
         {
             // Just copy
             map2Dwrite( nextDepthMap, i,j, hij,width );
         }
     }
 }

/**
 * Initialize the particle positions field
 **/
__global__ void initParticlePositionsCUDA( vec3* positionsMap, float minHeight, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        positionsMap[i].x = 0;
        positionsMap[i].y = minHeight - 1;
        positionsMap[i].z = 0;
    }
}

/**
 * Initialize the particle velocities field
 **/
__global__ void initParticleVelocitiesCUDA( vec3* velocitiesMap, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        velocitiesMap[i].x = 0;
        velocitiesMap[i].y = 0;
        velocitiesMap[i].z = 0;
    }
}

/**
 * Initialize the foam TTL field
 **/
__global__ void initFoamTTLCUDA( float* foamTTLMap, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        foamTTLMap[i] = 0;
    }
}

/**
 * Initialize the splash to foam field
 **/
__global__ void initSplashToFoamCUDA( float* splashToFoamMap, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        splashToFoamMap[i] = -1;
    }
}

/**
 * Update the particle positions and velocities fields
 **/
__global__ void updateParticleValuesCUDA( vec3* positionsMap, vec3* velocitiesMap, float minHeight, float accX, float accY, float accZ, float dt, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        //check if particle is active
        if(positionsMap[i].y >= minHeight){
            //update position vector
            positionsMap[i].x = positionsMap[i].x +
                    (velocitiesMap[i].x * dt) +
                    (accX * dt * dt);
            positionsMap[i].y = positionsMap[i].y +
                    (velocitiesMap[i].y * dt) +
                    (accY * dt * dt);
            positionsMap[i].z = positionsMap[i].z +
                    (velocitiesMap[i].z * dt) +
                    (accZ * dt * dt);

            //update velocity vector
            velocitiesMap[i].x = velocitiesMap[i].x + (accX * dt);
            velocitiesMap[i].y = velocitiesMap[i].y + (accY * dt);
            velocitiesMap[i].z = velocitiesMap[i].z + (accZ * dt);
        }
    }
}

/**
 * Update the foam particle fields
 **/
__global__ void updateFoamValuesCUDA( vec3* positionsMap, float* ttlsMap,
                                      float* heightMap, float* velUMap, float* velWMap,
                                      float width, float height, float uwidth, float uheight, float wwidth, float wheight,
                                      float minHeight, float dt, float halfDomain, float mdxInv, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        //check if particle is active
        if(ttlsMap[i] > 0){
            //take away a timestep
            ttlsMap[i] -= dt;

            //find grid positions x and z
            float lenX = (positionsMap[i].x + halfDomain) * mdxInv;
            float lenZ = (positionsMap[i].z + halfDomain) * mdxInv;
            int x = (int) cudaMin(width - 1, cudaMax(0.0, round(lenX)));
            int z = (int) cudaMin(height - 1, cudaMax(0.0, round(lenZ)));

            //get the velocities and height
            float hxz = map2Dread( heightMap, z, x, width );
            float uxz = map2Dread( velUMap, z, x, uwidth );
            float wxz = map2Dread( velWMap, z, x, wwidth );
            positionsMap[i].x += 1.0f * uxz * dt;
            positionsMap[i].y = hxz;
            positionsMap[i].z += 1.0f * wxz * dt;
        } else {
            positionsMap[i].y = minHeight - 1;
        }
    }
}

/**
 * Intersect the particles with the height and depth fields (splash, splash to foam)
 **/
__global__ void intersectParticleValuesCUDA( vec3* positionsMap, vec3* velocitiesMap, float* splashToFoamMap,
                                          float* heightMap, float* depthMap, float* velUMap, float* velWMap,
                                          const int width, const int height,
                                          const int uwidth, const int uheight,
                                          const int wwidth, const int wwheight,
                                          float minHeight, const float halfDomain, const float dx, const float mdxInv,
                                          float heightChange, const float Veff, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        //check if particle is active
        if(positionsMap[i].y >= minHeight){
            //find grid positions x and z
            float lenX = (positionsMap[i].x + halfDomain) * mdxInv;
            float lenZ = (positionsMap[i].z + halfDomain) * mdxInv;
            int x = (int) cudaMin(width - 1, cudaMax(0.0, round(lenX)));
            int z = (int) cudaMin(height - 1, cudaMax(0.0, round(lenZ)));

            //check if position y < heightMap
            float eta = map2Dread( heightMap, z, x, width );
            if(eta >= positionsMap[i].y){
                //update height
                float hxz = map2Dread( depthMap, z, x, width );
                map2Dwrite( depthMap, z, x, hxz + heightChange, width );

                //update velocities
                float uxz = map2Dread( velUMap, z, x, uwidth );
                float wxz = map2Dread( velWMap, z, x, wwidth );
                float term = hxz * dx * dx;
                map2Dwrite( velUMap, z, x, ((uxz * term) + (velocitiesMap[i].z * Veff)) / (term + Veff), uwidth );
                map2Dwrite( velWMap, z, x, ((wxz * term) + (velocitiesMap[i].x * Veff)) / (term + Veff), wwidth );

                positionsMap[i].y = minHeight - 1;
                splashToFoamMap[i] = 1;
            } else {
                splashToFoamMap[i] = -1;
            }
        } else {
            splashToFoamMap[i] = -1;
        }
    }
}

/**
 * Intersect the particles with the height and depth fields (spray, no splash to foam)
 **/
__global__ void intersectSprayParticleValuesCUDA( vec3* positionsMap, vec3* velocitiesMap,
                                          float* heightMap, float* depthMap, float* velUMap, float* velWMap,
                                          const int width, const int height,
                                          const int uwidth, const int uheight,
                                          const int wwidth, const int wwheight,
                                          float minHeight, const float halfDomain, const float dx, const float mdxInv,
                                          float heightChange, const float Veff, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        //check if particle is active
        if(positionsMap[i].y >= minHeight){
            //find grid positions x and z
            float lenX = (positionsMap[i].x + halfDomain) * mdxInv;
            float lenZ = (positionsMap[i].z + halfDomain) * mdxInv;
            int x = (int) cudaMin(width - 1, cudaMax(0.0, round(lenX)));
            int z = (int) cudaMin(height - 1, cudaMax(0.0, round(lenZ)));

            //check if position y < heightMap
            float eta = map2Dread( heightMap, z, x, width );
            if(eta >= positionsMap[i].y){
                //update height
                float hxz = map2Dread( depthMap, z, x, width );
                map2Dwrite( depthMap, z, x, hxz + heightChange, width );

                //update velocities
                float uxz = map2Dread( velUMap, z, x, uwidth );
                float wxz = map2Dread( velWMap, z, x, wwidth );
                float term = hxz * dx * dx;
                map2Dwrite( velUMap, z, x, ((uxz * term) + (velocitiesMap[i].z * Veff)) / (term + Veff), uwidth );
                map2Dwrite( velWMap, z, x, ((wxz * term) + (velocitiesMap[i].x * Veff)) / (term + Veff), wwidth );

                positionsMap[i].y = minHeight - 1;
            }
        }
    }
}

/**
 *  check for breaking waves
 */
__global__ void checkBreakingWavesCUDA( float* depthMap, float* prevDepthMap, float* heightMap, float* breakingWavesMap,
                                        const int width, const int height,
                                        const int uwidth, const int uheight,
                                        const int wwidth, const int wheight,
                                        const float condition1, const float condition2, const float condition3,
                                        const float mdxInv, const float dt )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;

    if( i >= 1 && i < height - 1 && j >= 1 && j < width - 1 )
    {
        map2Dwrite( breakingWavesMap, i, j, 0.0f, width );

        //eta terms
        float eta = map2Dread( heightMap, i, j, width );
        float etaIInc = map2Dread( heightMap, i + 1, j, width );
        float etaIDec = map2Dread( heightMap, i - 1, j, width );
        float etaJInc = map2Dread( heightMap, i, j + 1, width );
        float etaJDec = map2Dread( heightMap, i, j - 1, width );

        //additional eta terms (added by us, makes breaking waves nicer)
        float etaIIncJInc = map2Dread( heightMap, i + 1, j + 1, width );
        float etaIIncJDec = map2Dread( heightMap, i + 1, j - 1, width );
        float etaIDecJInc = map2Dread( heightMap, i - 1, j + 1, width );
        float etaIDecJDec = map2Dread( heightMap, i - 1, j - 1, width );

        //terms for first condition
        float firstTerm = etaIInc - eta;
        float secondTerm = etaIDec - eta;
        float thirdTerm = etaJInc - eta;
        float fourthTerm = etaJDec - eta;

        //eta gradient for first condition
        vec2 etaGradient;
        etaGradient.x = firstTerm;
        etaGradient.y = thirdTerm;
        if(fabs(secondTerm) > fabs(firstTerm)){
            etaGradient.x = secondTerm;
        }
        if(fabs(fourthTerm) > fabs(thirdTerm)){
            etaGradient.y = fourthTerm;
        }

        etaGradient.x *= mdxInv;
        etaGradient.y *= mdxInv;

        //magnitude for first condition
        float etaGradientMagnitude = sqrt((etaGradient.x * etaGradient.x) + (etaGradient.y * etaGradient.y));

        //check condition 1: steepness
        if(etaGradientMagnitude <= condition1){
            map2Dwrite( breakingWavesMap, i, j, 0.0f, width );
            return;
        }

        //depth change term
        float hij = map2Dread( depthMap, i, j, width );
        float hijPrev = map2Dread( prevDepthMap, i, j, width );
        float depthChange = (hij - hijPrev) / dt;

        //check condition 2: rising front
        if(depthChange <= condition2){
            map2Dwrite( breakingWavesMap, i, j, 0.0f, width );
            return;
        }

        //compute numerator for third condition
        //float numerator = etaIInc + etaIDec + etaJInc + etaJDec - (4 * eta);
        //float numerator = firstTerm + secondTerm + thirdTerm + fourthTerm;
        float numerator = etaIInc + etaIDec + etaJInc + etaJDec +
                etaIIncJInc + etaIIncJDec + etaIDecJInc + etaIDecJDec - (8 * eta);

        //check condition 3: top of wave
        if(numerator * mdxInv * mdxInv >= condition3){
            map2Dwrite( breakingWavesMap, i, j, 0.0f, width );
            return;
        }

        //write the depth change (used for vertical velocity of particles)
        map2Dwrite( breakingWavesMap, i, j, depthChange, width );
    }
}

/**
 *  Initialize breaking waves map
 */
__global__ void initBreakingWavesCUDA( float* breakingWavesMap, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;
    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        //initialize to 0
        map2Dwrite( breakingWavesMap, i, j, 0.0f, width );
    }
}

/**
 *  clamp the depth (min 0)
 */
__global__ void clampDepthCUDA( float* depthMap, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;

    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        float depth = cudaMax(0.f, map2Dread( depthMap, i, j, width ));
        map2Dwrite( depthMap, i, j, depth, width );
    }
}

/**
 * clamp the velocity (max velocity clamp)
 **/
__global__ void clampFieldCUDA( float* deviceMap, float velocityClamp, int width, int height )
{
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;
    if( i >= 0&& i < height&&j >= 0 && j < width )
    {
        float value = cudaMin(velocityClamp, map2Dread( deviceMap, i, j, width ));
        map2Dwrite( deviceMap, i, j, value, width );
    }
}

/**
 *  Initialize the sigma and gamma fields
 */
__global__ void initSigmaGammaCUDA( float* sigmaMap, float* gammaMap, const float dampeningRegion,
                                    const float quadraticA, const float quadraticB, const float quadraticC,
                                    const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;
    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        if(i < dampeningRegion || i >= height - dampeningRegion ||
                j < dampeningRegion || j >= width - dampeningRegion){
            //compute horizontal and vertical distances
            float iDistance = 0;
            float jDistance = 0;
            if(i < dampeningRegion){
                iDistance = dampeningRegion - i;
            } else if(i >= height - dampeningRegion){
                iDistance = i - (height - dampeningRegion - 1);
            }

            if(j < dampeningRegion){
                jDistance = dampeningRegion - j;
            } else if(j >= width - dampeningRegion){
                jDistance = j - (width - dampeningRegion - 1);
            }

            iDistance /= (float)dampeningRegion;
            jDistance /= (float)dampeningRegion;

            //distance
            float distance = sqrt((iDistance * iDistance) + (jDistance * jDistance));

            //quadratic function
            float value = (quadraticA * distance * distance) + (quadraticB * distance) + quadraticC;

            //initialize to value
            map2Dwrite( sigmaMap, i, j, value, width );
            map2Dwrite( gammaMap, i, j, value, width );
        } else {
            //initialize to 0
            map2Dwrite( sigmaMap, i, j, 0.0f, width );
            map2Dwrite( gammaMap, i, j, 0.0f, width );
        }
    }
}

/**
 *  Initialize the phi and psi fields
 */
__global__ void initPhiPsiCUDA( float* phiMap, float* psiMap, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;
    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        //initialize to 0
        map2Dwrite( phiMap, i, j, 0.0f, width );
        map2Dwrite( psiMap, i, j, 0.0f, width );
    }
}

/**
 *  Dampen the waves (dampen the depth field), update wave dampening data structures
 */
__global__ void dampenWavesCUDA( float* depthMap, float* heightMap, float* velUMap, float* velWMap,
                                 float* sigmaMap, float* gammaMap, float* phiMap, float* psiMap,
                                 float dampeningRegion, float hRest, float dt, float dxInv, float lambdaUpdate, float lambdaDecay,
                                 const int width, const int height, const int uwidth, const int uheight, const int wwidth, const int wheight )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;

    if( i >= 1 && i < height - 1 && j >= 1 && j < width - 1 )
    {
        if(i < dampeningRegion || i >= height - dampeningRegion ||
                j < dampeningRegion || j >= width - dampeningRegion){
            // current values
            float currH = map2Dread( heightMap, i, j, width );
            float currDepth = map2Dread( depthMap, i, j, width );

            float currVelU = map2Dread( velUMap, i, j, uwidth );
            float currVelUDec = map2Dread( velUMap, i - 1, j, uwidth );
            float currVelW = map2Dread( velWMap, i, j, wwidth );
            float currVelWDec = map2Dread( velWMap, i, j - 1, wwidth );

            float currSigma = map2Dread( sigmaMap, i, j, width );
            float currSigmaInc = map2Dread( sigmaMap, i + 1, j, width );
            float currGamma = map2Dread( gammaMap, i, j, width );
            float currGammaInc = map2Dread( gammaMap, i, j + 1, width );
            float currPhi = map2Dread( phiMap, i, j, width );
            float currPsi = map2Dread( psiMap, i, j, width );

            // Equation 10
            // h(i,j) += ((-sigma(i,j) * (h(i,j) - hRest)) + phi(i,j)) * delta_t
            // Equation 21
            // h(i,j) += ((-gamma(i,j) * (h(i,j) - hRest)) + psi(i,j)) * delta_t
            float eq10 = ((-currSigma * (currH - hRest)) + currPhi) * dt;
            float eq21 = ((-currGamma * (currH - hRest)) + currPsi) * dt;
            map2Dwrite( depthMap, i, j, currDepth + eq10 + eq21, width );

            // Equation 11
            // u(i+0.5,j) += -0.5 * (sigma(i+1,j) + sigma(i,j)) * u(i+0.5,j) * delta_t
            float eq11 = -0.5 * (currSigmaInc + currSigma) * currVelU * dt;
            map2Dwrite( velUMap, i, j, currVelU + eq11, uwidth);

            // Equation 22
            // w(i,j+0.5) += -0.5 * (gamma(i,j+1) + gamma(i,j)) * w(i,j+0.5) * delta_t
            float eq22 = -0.5 * (currGammaInc + currGamma) * currVelW * dt;
            map2Dwrite( velWMap, i, j, currVelW + eq22, wwidth);

            // Equation 12
            // phi(i,j) += -LAMBDA_UPDATE * sigma(i,j) * ((w(i,j+0.5) - w(i,j-0.5)) / delta_x) * delta_t
            // Equation 13
            // phi(i,j) *= LAMBDA_DECAY
            float eq12 = -lambdaUpdate * currSigma * (currVelW - currVelWDec) * dxInv * dt;
            map2Dwrite( phiMap, i, j, (currPhi + eq12) * lambdaDecay, width );

            // Equation 23
            // psi(i,j) += -LAMBDA_UPDATE * gamma(i,j) * ((u(i+0.5,j) - u(i-0.5,j)) / delta_x) * delta_t
            // Equation 24
            // psi(i,j) *= LAMBDA_DECAY
            float eq23 = -lambdaUpdate * currGamma * (currVelU - currVelUDec) * dxInv * dt;
            map2Dwrite( psiMap, i, j, (currPsi + eq23) * lambdaDecay, width );
        }
    }
}

 // Review passed
/**
 * @brief initGrid Initialize our grid
 * @param girdSize The gridSize
 * @param terrainMap The terrainMap from host
 */
void initGridGPU( const int hostGridSize, const int hostGridPaintSize, const float hostdx, const float halfdm, const float* hostTerrainMap )
{
    gridSize = hostGridSize;
    gridPaintSize = hostGridPaintSize;
    // Check the size
    assert( gridPaintSize == gridSize || gridPaintSize == gridSize+2 );
    halfDomain = halfdm;
    uwidth = gridSize + 1;
    uheight = gridSize;
    wwidth = gridSize;
    wheight = gridSize+1;
    mapdx = hostdx;
    mapdxInv = 1.f/mapdx;

    int width = gridSize;
    int height = gridSize;
    // Firstly backup the terrain's heightMap
    error = hipMalloc(&deviceTerrainMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceTerrainMap );
    error = hipMemcpy( deviceTerrainMap, hostTerrainMap, width*height*sizeof(float), hipMemcpyHostToDevice );
    checkCudaError( error );

    // Malloc heightMap
    error = hipMalloc(&deviceHeightMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceHeightMap );

    // Malloc depthMap
    error = hipMalloc(&deviceDepthMap,  width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceDepthMap );

    // Malloc depthMap
    error = hipMalloc(&devicePrevDepthMap,  width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( devicePrevDepthMap );

    // Malloc nextDepthMap
    error = hipMalloc(&deviceNextDepthMap,  width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceNextDepthMap );

    // Malloc breakingWavesMap
    error = hipMalloc(&deviceBreakingWavesMap, width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceNextDepthMap );

    width = gridPaintSize;
    height = gridPaintSize;
    // Malloc the normapMap
    error = hipMalloc(&devicePaintNormalMap,  width*height*sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( devicePaintNormalMap );
    // Malloc the paintMap
    error = hipMalloc(&devicePaintMap,  width*height*sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( devicePaintMap );

    width = uwidth;
    height = uheight;
    // Malloc velocityUMap
    error = hipMalloc(&deviceVelocityUMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceVelocityUMap );
    // Malloc nextVelocityUMap
    error = hipMalloc(&deviceNextVelocityUMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceNextVelocityUMap );

    width  =wwidth;
    height = wheight;
    // Malloc velocityWMap
    error = hipMalloc(&deviceVelocityWMap, width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceVelocityWMap );
    // Malloc velocityWMap
    error = hipMalloc(&deviceNextVelocityWMap, width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceNextVelocityWMap );

    // initialize the depth map
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    initDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceDepthMap,deviceTerrainMap,gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    // Copy the depth field to initialize the next depth map
    hipMemcpy(deviceNextDepthMap,deviceDepthMap,gridSize*gridSize*sizeof(float), hipMemcpyDeviceToDevice );

    // copy to initialize the previous depth map
    hipMemcpy(devicePrevDepthMap, deviceDepthMap, gridSize * gridSize * sizeof(float), hipMemcpyDeviceToDevice);

    //initialize breaking waves map
    initBreakingWavesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceBreakingWavesMap, gridSize, gridSize
                                                    );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    blockPerGridX = (gridPaintSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridPaintSize+ blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    // Initialize the normal map
    initPaintNormalCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   devicePaintNormalMap,gridPaintSize,gridPaintSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    // Initialize velocity U map
    blockPerGridX = (uwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    initFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityUMap, uwidth, uheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    // Copy the velocity U map to initialize next velocity U map
    hipMemcpy(deviceNextVelocityUMap,deviceVelocityUMap,(uwidth)*uheight*sizeof(float), hipMemcpyDeviceToDevice );


    //checkInitializedDeviceField( deviceVelocityUMap, gridSize+1, gridSize );
    //checkInitializedDeviceField( deviceNextVelocityUMap, gridSize+1, gridSize );


    // Initialize velocityW
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    initFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityWMap, wwidth, wheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    // Copy the velocity W map to initialize next velocity W map
    hipMemcpy(deviceNextVelocityWMap,deviceVelocityWMap,(wwidth)*wheight*sizeof(float), hipMemcpyDeviceToDevice );


    //checkInitializedDeviceField( deviceVelocityWMap, wwidth, wheight );
    //checkInitializedDeviceField( deviceNextVelocityWMap, wwidth, wheight );


    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceHeightMap,deviceDepthMap,deviceTerrainMap,
                                                   gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

   /* blockPerGridX = (gridPaintSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridPaintSize+ blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updatePaintCUDA<<<blocksPerGrid,threadsPerBlock>>>( devicePaintMap, deviceHeightMap, deviceDepthMap,
                                                        halfDomain, mapdx, gridPaintSize);
    error = hipDeviceSynchronize();
    checkCudaError(error);
    */
    blockPerGridX = (gridPaintSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridPaintSize+ blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    if( gridPaintSize == gridSize )
    {

        updatePaintCUDA<<<blocksPerGrid,threadsPerBlock>>>( devicePaintMap, deviceHeightMap, deviceDepthMap,
                                                            halfDomain, mapdx, gridPaintSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);

    }
    else if( gridPaintSize == gridSize + 2 )
    {
        updatePaintBoundCUDA<<<blocksPerGrid,threadsPerBlock>>>( devicePaintMap, deviceHeightMap, deviceDepthMap,
                                                            halfDomain, mapdx, gridPaintSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);
    }
    else
    {
        assert(0);
    }
}

// Review passed
/**
 * @brief updateFluidGPU Update function interface
 */
void updateFluidGPU( const float dt )
{
    /**
     * Advect the depth
     */
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    advectDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceNextDepthMap, deviceVelocityUMap,deviceVelocityWMap,
                     gridSize,gridSize,dt,mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    /**
     * Copy back the depth
     */
    hipMemcpy( deviceDepthMap, deviceNextDepthMap,
                gridSize*gridSize*sizeof(float), hipMemcpyDeviceToDevice);
    /**
     * Advect the velocity U
     */
    blockPerGridX = (uwidth+blockSizeX-1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY-1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    advectVelUCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityUMap, deviceNextVelocityUMap,
                                                       deviceVelocityWMap, uwidth,uheight, dt,mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Copy back the velocity U
     */
    hipMemcpy( deviceVelocityUMap, deviceNextVelocityUMap,
                uwidth*uheight*sizeof(float), hipMemcpyDeviceToDevice);
    /**
     * Advect the velocity W
     */
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    advectVelWCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityWMap, deviceNextVelocityWMap,
                    deviceVelocityUMap, wwidth, wheight, dt, mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Copy back the velocity W
     */
    hipMemcpy( deviceVelocityWMap, deviceNextVelocityWMap, wwidth*wheight*sizeof(float),hipMemcpyDeviceToDevice );


    /**
     * Update the depth
     */
    blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updateDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap,
                                                        deviceVelocityUMap, deviceVelocityWMap, gridSize,gridSize,dt,mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the height
     */
    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceHeightMap,
                                                         deviceDepthMap, deviceTerrainMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Apply the boundary
     */
    blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    applyBoundaryCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceHeightMap, deviceTerrainMap, gridSize,gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    overshootingReduction<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap,
                                                              deviceNextDepthMap, deviceHeightMap, mapdx, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the velocity U
     */
    blockPerGridX = (uwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updateVelUCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityUMap, deviceHeightMap, deviceDepthMap, uwidth, uheight, dt, mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the velocity W
     */
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updateVelWCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityWMap, deviceHeightMap, deviceDepthMap, wwidth, wheight, dt, mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Apply the boundary
     */
    blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    applyBoundaryCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceHeightMap, deviceTerrainMap, gridSize,gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    overshootingReduction<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap,
                                                              deviceNextDepthMap, deviceHeightMap, mapdx, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    /**
     * Copy back the buffer into deviceDepthMap
     */
    hipMemcpy( deviceDepthMap, deviceNextDepthMap, gridSize*gridSize*sizeof(float),hipMemcpyDeviceToDevice );

    /**
     * Apply boundary again
     **/
    applyBoundaryCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceHeightMap, deviceTerrainMap, gridSize,gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * update the height map
     */
    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Compute he normal map
     */
    blockPerGridX = (gridPaintSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridPaintSize+ blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    computePaintNormalCUDA<<<blocksPerGrid,threadsPerBlock>>>( devicePaintNormalMap, devicePaintMap,
                                                          gridPaintSize, gridPaintSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the paint map
     */
    blockPerGridX = (gridPaintSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridPaintSize+ blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    if( gridPaintSize == gridSize )
    {

        updatePaintCUDA<<<blocksPerGrid,threadsPerBlock>>>( devicePaintMap, deviceHeightMap, deviceDepthMap,
                                                            halfDomain, mapdx, gridPaintSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);

    }
    else if( gridPaintSize == gridSize + 2 )
    {
        updatePaintBoundCUDA<<<blocksPerGrid,threadsPerBlock>>>( devicePaintMap, deviceHeightMap, deviceDepthMap,
                                                            halfDomain, mapdx, gridPaintSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);
    }
    else
    {
        assert(0);
    }
}

// Review passed
/**
 * @brief addDropGPU Add drop interface
 * @param posX The x coordinate
 * @param posZ The y coordinate
 * @param radius The radius
 * @param h The height added
 */
void addDropGPU(const int posX, const int posZ, const int radius, const float h )
{
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    addDropCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceDepthMap,posX,posZ,radius,h,gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceHeightMap,deviceDepthMap,deviceTerrainMap,
                                                   gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void addSideWaveGPU(const int sideIndex, const int sideLength, const float h )
{
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    addSideWaveCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceDepthMap, sideLength, sideIndex, h, gridSize, gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceHeightMap,deviceDepthMap,deviceTerrainMap,
                                                   gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

/**
 * @brief destroyCUDAmem Destroy the cuda memory
 */
void destroyGPUmem()
{
    hipFree( devicePaintMap );
    hipFree( devicePaintNormalMap );
    hipFree( deviceTerrainMap );
    hipFree( deviceHeightMap );
    hipFree( deviceDepthMap );
    hipFree( devicePrevDepthMap );
    hipFree( deviceVelocityUMap );
    hipFree( deviceVelocityWMap );
    hipFree( deviceNextDepthMap );
    hipFree( deviceNextVelocityUMap );
    hipFree( deviceNextVelocityWMap );

    hipFree( deviceParticlePositionsArray );
    hipFree( deviceParticleVelocitiesArray );
    hipFree( deviceSprayPositionsArray );
    hipFree( deviceSprayVelocitiesArray );
    hipFree( deviceFoamPositionsArray );
    hipFree( deviceFoamTTLArray );
    hipFree( deviceSplashToFoamArray );

    hipFree( deviceBreakingWavesMap );

    hipDeviceReset();
    hipDeviceReset();
}

// Review passed
/**
 * @brief copyback After each update we need to copy back the map
 * @param host The target host map
 */
void copybackGPU(FieldType type, float* hostMap  )
{
    switch( type )
    {
    case HEIGHT:
    {
        error = hipMemcpy( hostMap, deviceHeightMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case DEPTH:
    {
        error = hipMemcpy( hostMap, deviceDepthMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case NORMAL:
    {
        error = hipMemcpy( hostMap, devicePaintNormalMap, gridPaintSize*gridPaintSize*sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    case PAINT:
    {
         error = hipMemcpy( hostMap, devicePaintMap, gridPaintSize*gridPaintSize*sizeof(vec3), hipMemcpyDeviceToHost);
         break;
    }
    case PARTICLE_POSITIONS:
    {
        error = hipMemcpy( hostMap, deviceParticlePositionsArray, deviceNumSplashParticles * sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    case PARTICLE_VELOCITIES:
    {
        error = hipMemcpy( hostMap, deviceParticleVelocitiesArray, deviceNumSplashParticles * sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    case SPRAY_POSITIONS:
    {
        error = hipMemcpy( hostMap, deviceSprayPositionsArray, deviceNumSprayParticles * sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    case SPRAY_VELOCITIES:
    {
        error = hipMemcpy( hostMap, deviceSprayVelocitiesArray, deviceNumSprayParticles * sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    case FOAM_POSITIONS:
    {
        error = hipMemcpy( hostMap, deviceFoamPositionsArray, deviceNumFoamParticles * sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    case FOAM_TTLS:
    {
        error = hipMemcpy( hostMap, deviceFoamTTLArray, deviceNumFoamParticles * sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case SPLASH_TO_FOAM:
    {
        error = hipMemcpy( hostMap, deviceSplashToFoamArray, deviceNumSplashParticles * sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case SIGMA:
    {
        error = hipMemcpy( hostMap, deviceSigmaMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case GAMMA:
    {
        error = hipMemcpy( hostMap, deviceGammaMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case PHI:
    {
        error = hipMemcpy( hostMap, devicePhiMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case PSI:
    {
        error = hipMemcpy( hostMap, devicePsiMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case BREAKING_WAVES:
    {
        error = hipMemcpy( hostMap, deviceBreakingWavesMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost );
        break;
    }
    case VEL_U:
    {
        error = hipMemcpy( hostMap, deviceVelocityUMap, uwidth*uheight*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case VEL_W:
    {
        error = hipMemcpy( hostMap, deviceVelocityWMap, wwidth*wheight*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    default:
    {
        assert(0);
        break;
    }
    }
    checkCudaError( error );
}

/**
 * @brief findSupportGPU Find supported CUDA device counts
 * @return True if device count is not zero
 */
bool findSupportDevice()
{
       int deviceCount = 0;

       hipError_t error_id = hipGetDeviceCount(&deviceCount);

       if (error_id != hipSuccess)
       {
           printf("hipGetDeviceCount returned error code: %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
           printf("> FAILED %s sample finished, exiting...\n" );
           // I turn on the exit, it will never return
           exit(EXIT_FAILURE);
           return false;
       }
       if (deviceCount == 0)
       {
           printf("> There are no device(s) supporting CUDA\n");
           return false;
       }
       else
       {
           printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
           return true;
       }
}

void initParticlesGPU(const float minHeight, const int numSplashParticles, const int numSprayParticles, const int numFoamParticles){
    deviceNumSplashParticles = numSplashParticles;
    deviceNumSprayParticles = numSprayParticles;
    deviceNumFoamParticles = numFoamParticles;

    //malloc the arrays
    //splash
    error = hipMalloc(&deviceParticlePositionsArray,  deviceNumSplashParticles * sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( deviceParticlePositionsArray );

    error = hipMalloc(&deviceParticleVelocitiesArray,  deviceNumSplashParticles * sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( deviceParticleVelocitiesArray );

    //spray
    error = hipMalloc(&deviceSprayPositionsArray,  deviceNumSprayParticles * sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( deviceSprayPositionsArray );

    error = hipMalloc(&deviceSprayVelocitiesArray,  deviceNumSprayParticles * sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( deviceSprayVelocitiesArray );

    //foam
    error = hipMalloc(&deviceFoamPositionsArray,  deviceNumFoamParticles * sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( deviceFoamPositionsArray );

    error = hipMalloc(&deviceFoamTTLArray,  deviceNumFoamParticles * sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceFoamTTLArray );

    //splash to foam array
    error = hipMalloc(&deviceSplashToFoamArray,  deviceNumSplashParticles * sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceSplashToFoamArray );

    //set up the iterator properties
    int threadsPerBlock = 256;
    int blocksPerGrid = (deviceNumSplashParticles + threadsPerBlock - 1) / threadsPerBlock;

    //splash
    //initialize positions
    initParticlePositionsCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceParticlePositionsArray,
                                                    minHeight, deviceNumSplashParticles
                                                    );

    //initialize velocities
    initParticleVelocitiesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceParticleVelocitiesArray,
                                                    deviceNumSplashParticles
                                                    );

    //initialize splash to foam
    initSplashToFoamCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceSplashToFoamArray,
                                                    deviceNumSplashParticles
                                                    );

    //spray
    blocksPerGrid = (deviceNumSprayParticles + threadsPerBlock - 1) / threadsPerBlock;

    //initialize positions
    initParticlePositionsCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceSprayPositionsArray,
                                                    minHeight, deviceNumSprayParticles
                                                    );

    //initialize velocities
    initParticleVelocitiesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceSprayVelocitiesArray,
                                                    deviceNumSprayParticles
                                                    );

    //foam
    blocksPerGrid = (deviceNumFoamParticles + threadsPerBlock - 1) / threadsPerBlock;

    //initialize positions
    initParticlePositionsCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceFoamPositionsArray,
                                                    minHeight, deviceNumFoamParticles
                                                    );

    //initialize foam TTL
    initFoamTTLCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceFoamTTLArray,
                                                    deviceNumFoamParticles
                                                    );

    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void updateParticlesGPU( const float minHeight, const float dt, const float halfDomain, const float mdxInv, const float accX, const float accY, const float accZ ){
    //set up the iterator properties
    int threadsPerBlock = 256;
    int blocksPerGrid = (deviceNumSplashParticles + threadsPerBlock - 1) / threadsPerBlock;

    //splash
    //update positions and velocities
    updateParticleValuesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceParticlePositionsArray,
                                                    deviceParticleVelocitiesArray,
                                                    minHeight, accX, accY, accZ, dt,
                                                    deviceNumSplashParticles
                                                    );

    //spray
    blocksPerGrid = (deviceNumSprayParticles + threadsPerBlock - 1) / threadsPerBlock;
    updateParticleValuesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceSprayPositionsArray,
                                                    deviceSprayVelocitiesArray,
                                                    minHeight, accX, accY, accZ, dt,
                                                    deviceNumSprayParticles
                                                    );

    //foam
    blocksPerGrid = (deviceNumFoamParticles + threadsPerBlock - 1) / threadsPerBlock;
    updateFoamValuesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceFoamPositionsArray, deviceFoamTTLArray,
                                                    deviceHeightMap, deviceVelocityUMap, deviceVelocityWMap,
                                                    gridSize, gridSize, uwidth, uheight, wwidth, wheight,
                                                    minHeight, dt, halfDomain, mdxInv, deviceNumFoamParticles
                                                    );

    //error check
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void intersectParticlesGPU( const float minHeight, const float halfDomain, const float mdx, const float mdxInv,
                            const float splashVeff, const float splashHeightChange, const float sprayVeff, const float sprayHeightChange ){
    //set up the iterator properties
    int threadsPerBlock = 256;
    int blocksPerGrid = (deviceNumSplashParticles + threadsPerBlock - 1) / threadsPerBlock;

    //splash
    //intersect particles and velocities
    intersectParticleValuesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceParticlePositionsArray, deviceParticleVelocitiesArray,
                                                    deviceSplashToFoamArray,
                                                    deviceHeightMap, deviceDepthMap, deviceVelocityUMap, deviceVelocityWMap,
                                                    gridSize, gridSize,
                                                    uwidth, uheight,
                                                    wwidth, wheight,
                                                    minHeight, halfDomain, mdx, mdxInv, splashHeightChange, splashVeff, deviceNumSplashParticles
                                                    );
    //spray
    blocksPerGrid = (deviceNumSprayParticles + threadsPerBlock - 1) / threadsPerBlock;
    intersectSprayParticleValuesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceSprayPositionsArray, deviceSprayVelocitiesArray,
                                                    deviceHeightMap, deviceDepthMap, deviceVelocityUMap, deviceVelocityWMap,
                                                    gridSize, gridSize,
                                                    uwidth, uheight,
                                                    wwidth, wheight,
                                                    minHeight, halfDomain, mdx, mdxInv, sprayHeightChange, sprayVeff, deviceNumSplashParticles
                                                    );

    //set up grid iterator
    dim3 threadsPerBlock2(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid2(blockPerGridX,blockPerGridY);

    //update height field
    updateHeightCUDA<<<blocksPerGrid2, threadsPerBlock2>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap,
                                 gridSize, gridSize );

    //error check
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void inputParticlesGPU( const float *particlePositions, const float *particleVelocities ){
    //copy over
    error = hipMemcpy( deviceParticlePositionsArray, particlePositions, deviceNumSplashParticles * sizeof(vec3), hipMemcpyHostToDevice );
    checkCudaError( error );

    error = hipMemcpy( deviceParticleVelocitiesArray, particleVelocities, deviceNumSplashParticles * sizeof(vec3), hipMemcpyHostToDevice );
    checkCudaError( error );
}

void inputSprayParticlesGPU( const float *particlePositions, const float *particleVelocities ){
    //copy over
    error = hipMemcpy( deviceSprayPositionsArray, particlePositions, deviceNumSprayParticles * sizeof(vec3), hipMemcpyHostToDevice );
    checkCudaError( error );

    error = hipMemcpy( deviceSprayVelocitiesArray, particleVelocities, deviceNumSprayParticles * sizeof(vec3), hipMemcpyHostToDevice );
    checkCudaError( error );
}

void inputFoamParticlesGPU( const float *particlePositions, const float *ttlArray ){
    //copy over
    error = hipMemcpy( deviceFoamPositionsArray, particlePositions, deviceNumFoamParticles * sizeof(vec3), hipMemcpyHostToDevice );
    checkCudaError( error );

    error = hipMemcpy( deviceFoamTTLArray, ttlArray, deviceNumFoamParticles * sizeof(float), hipMemcpyHostToDevice );
    checkCudaError( error );
}

void checkBreakingWavesGPU( const float condition1, const float condition2, const float condition3,
                            const float mdxInv, const float dt ){
    // iterate over grids, make above 0
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);

    checkBreakingWavesCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                    deviceDepthMap, devicePrevDepthMap, deviceHeightMap, deviceBreakingWavesMap,
                                                    gridSize, gridSize,
                                                    uwidth, uheight,
                                                    wwidth, wheight,
                                                    condition1, condition2, condition3,
                                                    mdxInv, dt
                                                    );
    //update height field
    updateHeightCUDA<<<blocksPerGrid, threadsPerBlock>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap,
                                 gridSize, gridSize );

    //store the current depth as the previous depth for the next timestep
    hipMemcpy(devicePrevDepthMap, deviceDepthMap, gridSize * gridSize * sizeof(float), hipMemcpyDeviceToDevice);

    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void clampFieldsGPU( const float velocityClamp ){
    // iterate over depth field, make above 0
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    //clamp depth, min value is 0
    clampDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceDepthMap, gridSize, gridSize
                                                   );
    //update height field
    updateHeightCUDA<<<blocksPerGrid, threadsPerBlock>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap,
                                 gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    //iterate over velocity U, make below velocity clamp
    blockPerGridX = (uwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    //clamp velocity u, max value is velocityClamp
    clampFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityUMap, velocityClamp, uwidth, uheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    //iterate over velocity W, make below velocity clamp
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    //clamp velocity w, max value is velocityClamp
    clampFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityWMap, velocityClamp, wwidth, wheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void initDampeningFieldsGPU( const int sizeDampeningRegion, const float quadraticA, const float quadraticB, const float quadraticC ){
    //set size of dampening region
    deviceDampeningRegion = sizeDampeningRegion;

    //get width and height
    int width = gridSize;
    int height = gridSize;

    //initialize arrays
    //initialize sigma
    error = hipMalloc(&deviceSigmaMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceSigmaMap );

    //initialize gamma
    error = hipMalloc(&deviceGammaMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceGammaMap );

    //initialize phi
    error = hipMalloc(&devicePhiMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( devicePhiMap );

    //initialize psi
    error = hipMalloc(&devicePsiMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( devicePsiMap );

    //fill arrays
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);

    //fill sigma and gamma
    initSigmaGammaCUDA<<<blocksPerGrid, threadsPerBlock>>>( deviceSigmaMap, deviceGammaMap,
                                                    deviceDampeningRegion, quadraticA, quadraticB, quadraticC,
                                                    gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    //fill phi and psi
    initPhiPsiCUDA<<<blocksPerGrid, threadsPerBlock>>>( devicePhiMap, devicePsiMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void dampenWavesGPU( const float hRest, const float dt, const float dxInv, const float lambdaUpdate, const float lambdaDecay ){
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);

    //dampen waves
    dampenWavesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                    deviceDepthMap, deviceHeightMap, deviceVelocityUMap, deviceVelocityWMap,
                                    deviceSigmaMap, deviceGammaMap, devicePhiMap, devicePsiMap,
                                    deviceDampeningRegion, hRest, dt, dxInv, lambdaUpdate, lambdaDecay,
                                    gridSize, gridSize, uwidth, uheight, wwidth, wheight
                                    );
    //update height field
    updateHeightCUDA<<<blocksPerGrid, threadsPerBlock>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap,
                                 gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void inputDepthGPU( const float* newDepthField ){
    //copy over
    error = hipMemcpy( deviceDepthMap, newDepthField, gridSize * gridSize * sizeof(float), hipMemcpyHostToDevice );
    checkCudaError( error );

    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);

    //update height field
    updateHeightCUDA<<<blocksPerGrid, threadsPerBlock>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap,
                                 gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

#endif
