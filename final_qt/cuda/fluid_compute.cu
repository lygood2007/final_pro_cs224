#include "hip/hip_runtime.h"
/** fluid_compute.cu
 ** Brief: Deal with all the computation here
 ** Project: large-scale fluids
 ** Date: 04/10/2013
 ** Member: Scott, Hobarts, Yan Li
 **/

#ifndef FLUID_COMPUTE_CU
#define FLUID_COMPUTE_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include "fluid_global.h"
// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

extern "C"
{
void initGridGPU( const int hostGridSize, const float hostdx, const float* hostTerrainMap );
void copybackGPU(FieldType type, float* hostMap  );
void destroyGPUmem();
void addDropGPU(const int posX, const int posZ, const int radius, const float h );
void advectGPU(const float dt);
void updateFluidGPU( const float dt );
bool findSupportDevice();
}

__host__ __device__ inline float cudaMax( float a, float b )
{
    return (a>b)?(a):(b);
}

__host__ __device__ inline float cudaMin( float a, float b )
{
    return (a<b)?(a):(b);
}


// The 2D vector structure for GPU computing
 struct vec2
{
    union
    {
        struct {float x,y; };
        float xy[2];
    };
};

 struct vec3
{
    union
    {
        struct {float x,y,z; };
        float xyz[3];
    };
};

const int blockSizeX = 4;
const int blockSizeY = 4;

vec3* deviceNormalMap; // Normal map for GPU
float* deviceTerrainMap; // Terrain map for GPU
float* deviceHeightMap; // Height map for GPU
float* deviceDepthMap; // Depth map for GPU
float* deviceVelocityUMap; // VelocityU map for GPU
float* deviceVelocityWMap; // VelocityW map for GPU

float* deviceNextDepthMap; // Temp buffer for storing next depth map
float* deviceNextVelocityUMap; // Temp buffer for storing next velocity U map
float* deviceNextVelocityWMap; // Temp buffer for storing next velocity W map

/**
 * pitches for the maps above
 */
// Error
hipError_t error;

// The grid size for heightmap, depthmap, terrainmap
int gridSize;
// The width for velocity u
int uwidth;
// The height for veloctiy u
int uheight;
// The width for velocity w
int wwidth;
// The height for veloctiy w
int wheight;

// dx
float mapdx;
// dxInv
float mapdxInv;

// Forward decaration
void initGridGPU( const int hostGirdSize, const float* hostTerrainMap );
void addDropGPU(const int posX, const int posZ, const int radius, const float h );
void copybackGPU(float* hostHeightMap );
void destroyGPU();
void advectGPU(const float dt);
void updateFluidGPU( const float dt );


void checkInitializedDeviceField( float* device, int width, int height )
{
    float* host = (float*)malloc(width*height*sizeof(float));
    hipMemcpy( host, device, width*height*sizeof(float),hipMemcpyDeviceToHost);

    for( int i = 0; i < width*height; i++ )
    {
        if( host[i] != 0.f )
        {
            assert(0);
        }
    }
    free(host);
}

void  checkCudaError( hipError_t error )
{
    if( error != hipSuccess )
    {
        //cout <<"CUDA error code: "<<hipGetErrorString(error);
        printf( "CUDA error code: %s\n",hipGetErrorString(error) );
    }
}
/**
 * Check if the pointer is null, if it's null, exit the program
 */
template <class T>
void check1DNotNull(T* array )
{
    if( array == NULL )
    {
        printf("Wrong pointer\n");
    }
}

/**
 * Initialize a vec2
 */
__host__ __device__ inline vec2 initVec2( float x, float y )
{
    vec2 result;
    result.x = x;result.y = y;
    return result;
}

/**
 *  Initialize a vec3
 */
__host__ __device__ inline vec3 initVec3( float x, float y, float z )
{
    vec3 result;
    result.x = x;result.y = y; result.z = z;
    return result;
}

// Review passed
/**
* Compute the cross product of two vectors
*/
__host__ __device__ inline vec3 cross( const vec3 v1, const vec3 v2 )
{
    vec3 result = initVec3(v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z, v1.x * v2.y - v1.y * v2.x);
    return result;
}

// Review passed
/**
 * Compute the normalized vector
 */
__host__ __device__ inline vec3 normalize( const vec3 v )
{
    float d = sqrt(v.x*v.x + v.y*v.y + v.z*v.z );
    return initVec3( v.x/d,v.y/d, v.z/d );
}

// Review passed
/**
 * Functions for getting data from 2D array in GPU
 */
__host__ __device__ inline float map2Dread( const float* map, int i, int j, int width )
{
    return map[i*width+ j];
}
// Review passed
/**
 * Functions for writting data from 2D array in GPU
 */
__host__ __device__ inline void map2Dwrite( float* map, int i, int j, float value, int width )
{
    map[i*width + j] = value;
}

// Review passed
/**
 *  Initialize the depth
 */
__global__ void initDepthCUDA( float* depthMap, const float* terrainMap, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;

    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        float depth = cudaMax(0.f,defaultHeight - map2Dread( terrainMap, i,j, width ));
        map2Dwrite( depthMap,i,j,depth,width );
    }
}

// Review passed
/**
 * Update the height field by plusing the depth and terrain
 */
__global__ void updateHeightCUDA( float*heightMap, const float* depthMap, const float* terrainMap,
                              const int width, const int height )
{
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        float depth = map2Dread( depthMap,i,j,width );
        float terrainHeight = map2Dread( terrainMap, i,j,width );
        float h = depth + terrainHeight;
        map2Dwrite( heightMap, i,j, h ,width);
    }
}

// Review passed
/**
 * Add drop to specified rectangular region
 */
__global__ void addDropCUDA( float* depthMap, const int posX, const int posZ, const int radius,
                             const float h, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    if( i>= cudaMax(posZ-radius,0) && i < cudaMin(posZ+radius+1,height)
            && j >= cudaMax(posX-radius,0)&&j < cudaMin(posX+radius+1,width)
            )
    {
        float newH = cudaMin(map2Dread(depthMap,i,j,width)+h,maxHeight );
        map2Dwrite( depthMap, i,j, newH, width );
    }
}

// Review passed
/**
 * bilinear interpolation
 */
 __host__ __device__ float bilinearIerp( const float* vec, float x, float z, const int width, const int height )
 {
     if( x < 0 )
         x = 0.f;
     if( z < 0 )
         z = 0.f;
     if( x > width - 1 )
         x = width - 1;
     if( z > height - 1 )
         z = height -1;

     const int X = (int)x;
     const int Y = (int)z;
     const float s1 = x - X;
     const float s0 = 1.f - s1;
     const float t1 = z - Y;
     const float t0 = 1.f-t1;
     float e1, e2, e3,e4;
     e1 = e2 = e3 = e4 = 0;
     //e1 = vec[Y*width+X];
     e1 = map2Dread(vec,Y,X,width);
     if( Y+1 <= height- 1 )
     {
      //   e2 = vec[(Y+1)*width + X];
         e2 = map2Dread( vec, Y+1,X,width);
     }
     if( X +1 <= width -1 )
     {
        // e3 = vec[Y*width + X+1];
         e3 = map2Dread( vec, Y, X+1, width );
     }
     if( Y+1 <= height - 1 && X + 1 <= width - 1)
     {
      //   e4 = vec[(Y+1)*width + X+1];
         e4 = map2Dread( vec, Y+1,x+1,width );
     }

     float result = s0*(t0*e1 + t1*e2 )+
             s1*(t0*e3  + t1*e4 );

     return  result;
 }

 // Review passed
 /**
  * Advection: depth
  **/
 __global__ void advectDepthCUDA( const float* depthMap, float* nextDepthMap, const float* velUMap, const float* velWMap,
                              const int width, const int height, const float dt, const float dxInv)
{
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height -1 && j >= 1 && j < width - 1 )
     {
         // Be careful about the width of velU
         float uw = width + 1;
         float u = 0.5*(map2Dread(velUMap,i,j,uw) + map2Dread( velUMap,i,j+1,uw) );
         float w = 0.5*(map2Dread(velWMap,i,j,width) + map2Dread( velWMap,i+1,j,width) );

         float curPosX = (float)j;
         float curPosY = (float)i;
         float prev_x = curPosX - u*dt*dxInv;
         float prev_z = curPosY - w*dt*dxInv;
         map2Dwrite(nextDepthMap, i,j, bilinearIerp( depthMap,prev_x,prev_z,width,height ), width);
     }
 }

 // Review passed
 /**
  * Advection: velocity U
  */
 __global__ void advectVelUCUDA( const float* velUMap, float* nextVelUMap, const float* velWMap,
                            const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height - 1&&j >= 1 && j < width - 1)
     {
         float ww = width - 1;
        float u = map2Dread( velUMap, i,j,width );
        float w = 0.25*(map2Dread(velWMap,i,j, ww ) + map2Dread( velWMap, i,j-1,ww) + map2Dread( velWMap,i+1,j-1,ww) + map2Dread( velWMap, i+1,j,ww ) );
        float curPosX = (float)j;
        float curPosY = (float)i;
        float prev_x = curPosX - u*dt*dxInv;
        float prev_z = curPosY - w*dt*dxInv;
        map2Dwrite(nextVelUMap, i,j, bilinearIerp(  velUMap,prev_x,prev_z,width,height ), width);
     }
 }

 // Review passed
 /**
  *  Advection: velocity W
  */
 __global__ void advectVelWCUDA( const float* velWMap, float* nextVelWMap, const float* velUMap,
                            const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height - 1&&j >= 1 && j < width - 1)
     {
        float uw = width + 1;
        float u = 0.25*(map2Dread( velUMap,i,j,uw) + map2Dread(velUMap,i,j+1,uw) + map2Dread(velUMap,i-1,j+1,uw) + map2Dread(velUMap, i-1,j,uw) );
        float w = map2Dread( velWMap,i,j,width );
        float curPosX = (float)j;
        float curPosY = (float)i;
        float prev_x = curPosX - u*dt*dxInv;
        float prev_z = curPosY - w*dt*dxInv;
        map2Dwrite(nextVelWMap,i,j,bilinearIerp(  velWMap,prev_x,prev_z,width,height ), width);
     }
 }

 // Review passed
 /**
  * Update the depth field
  **/
 __global__ void updateDepthCUDA(float* depthMap, const float* velUMap, const float* velWMap,
                                 const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height-1 && j >= 1 && j < width-1)
     {
            float decay = 1.f;
            float dep = map2Dread(depthMap,i,j,width);
            float dh = -decay*dep*dxInv*( (map2Dread(velUMap,i,j+1,width+1) - map2Dread(velUMap,i,j,width+1))
                                          + (map2Dread(velWMap,i+1,j,width) - map2Dread(velWMap,i,j,width)) );
            map2Dwrite(depthMap, i, j, dh*dt+dep, width );
     }
 }

 // Review passed
 /**
  * Update the velocity U field
  */
 __global__ void updateVelUCUDA( float* velUMap, const float* heightMap,
                                 const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >=1 && i < height - 1 && j >= 2 && j < width - 2 )
     {
         // The width of heightmap is 1 smaller than the width of velocity U
        float h1 = map2Dread( heightMap, i,j, width - 1);
        float h2 = map2Dread(  heightMap, i,j-1, width-1 );

        // Read the origin value from velUMap
        float vel = map2Dread( velUMap, i,j, width );
        float dv = GRAVITY*dt*dxInv*(h1-h2);

        // Add
        map2Dwrite( velUMap, i,j,vel+dv, width);
     }
     else
     {
         // for the bounday, we set the velocity to zero, using the neuman boundary condition
        map2Dwrite( velUMap, i,j,0.f, width);
     }
 }

 // Review passed
 /**
  * Update the velocity W field
  */
 __global__ void updateVelWCUDA( float* velWMap, const float* heightMap,
                                 const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 2 && i < height - 2 && j >= 1 && j < width - 1 )
     {
         float h1 = map2Dread( heightMap, i,j, width );
         float h2 = map2Dread(  heightMap, i-1,j, width );

         float vel = map2Dread( velWMap, i,j, width );
         float dv = GRAVITY*dt*dxInv*(h1-h2);
         map2Dwrite( velWMap, i,j,vel+dv, width );
     }
     else
     {
         // for the bounday, we set the velocity to zero, using the neuman boundary condition
        map2Dwrite( velWMap, i,j,0.f, width);
     }
 }

 // Review passed
 /**
  * Apply the boundary condition
  **/
 __global__ void applyBoundaryCUDA( float* depthMap, const float* heightMap, const float* terrainMap,
                                    const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
    float value;
    // Fix the boundary
    /*if( i == 0 || i == height-1|| j == 0 ||  j == width-1 )
    {
        map2Dwrite( depthMap, i,j, 0, width );
    }*/
     if( j == 0 && i !=  0 && i != height-1)
     {
         value = max(0.f, map2Dread( heightMap, i,1,width) - map2Dread( terrainMap, i,j,width ) );
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if( j== width - 1&& i !=0 && i != height - 1 )
     {
         value = max( 0.f, map2Dread( heightMap, i, width - 2, width ) - map2Dread( terrainMap, i, j,width) );
        map2Dwrite( depthMap, i, j, value, width );
        return;
     }

     if( i == 0&& j != 0 && j != width -1 )
     {
         value = max(0.f, map2Dread( heightMap, 1, j, width) - map2Dread( terrainMap, i, j,width ));
         map2Dwrite( depthMap, i, j, value, width );
         return;
     }
     else if( i == height - 1&& j != width - 1 && j != 0)
     {
         value = max(0.f, map2Dread( heightMap, height - 2, j, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i, j, value, width );
         return;
     }

     // Deal with the four courner, is there a way to simplify this? This function is too long!
     if( i== 0 && j == 0 )
     {
         value = max(0.f, map2Dread( heightMap, 1, 1, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if( i==0 && j == width - 1 )
     {
         value = max(0.f, map2Dread( heightMap, 1, width-2, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if ( i == height - 1 && j == 0 )
     {
         value = max(0.f, map2Dread( heightMap, height - 2, 1, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if( i == height - 1 && j == width - 1 )
     {
         value = max(0.f, map2Dread( heightMap, height - 2, width-2, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
 }

 // Review passed
 /**
  * A general function to initialize the field to be all zero( Cannot use memset because we are using float )
  **/
 __global__ void initFieldCUDA( float* deviceMap, int width, int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
         map2Dwrite( deviceMap, i, j, 0.f, width );
     }
 }

/**
 * Initialize the normal field
 **/
 __global__ void initNormalCUDA( vec3* normalMap, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
         const int index = i*width + j;
         normalMap[index].x = 0;
         normalMap[index].y = 1;
         normalMap[index].z = 0;
     }
 }

 /**
  * compute the normals
  **/
 __global__ void computeNormalCUDA( vec3* normalMap, const float* heightMap, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
            int numNeighbours = 0;

            vec3 offset[8];
            vec2 coords[8];
            vec2 neighbours[8];

            coords[0] = initVec2(i,     j - 1);
            coords[1] = initVec2(i + 1, j - 1);
            coords[2] = initVec2(i + 1, j);
            coords[3] = initVec2(i + 1, j + 1);
            coords[4] = initVec2(i,     j + 1);
            coords[5] = initVec2(i - 1, j + 1);
            coords[6] = initVec2(i - 1, j);
            coords[7] = initVec2(i - 1, j - 1);
            int m;
            for( m = 0; m < 8; m++ )
            {
                if( coords[m].x < 0 || coords[m].y < 0 || coords[m].x > height- 1 || coords[m].y > width- 1 )
                    continue;
                neighbours[numNeighbours] = coords[m];
                numNeighbours++;
            }

            for( m = 0; m < numNeighbours; m++ )
            {
                offset[m].x = neighbours[m].y - j;
                offset[m].z = neighbours[m].x - i;
                offset[m].y = map2Dread(heightMap,neighbours[m].x,neighbours[m].y,width) - map2Dread( heightMap, i,j,width );
            }

            vec3 sum = initVec3(0.f,0.f,0.f);
            for( m = 0; m < numNeighbours; m++ )
            {
                vec3 tmp;
                if( m+1 == numNeighbours )
                    tmp = cross( offset[m],offset[0]);
                else
                    tmp = cross( offset[m],offset[m+1]);
                sum.x += tmp.x;
                sum.y += tmp.y;
                sum.z += tmp.z;
            }
            vec3 result = normalize( sum );
           const int index = i*width+j;
           normalMap[index].x = result.x; normalMap[index].y = result.y; normalMap[index].z = result.z;
     }
 }

 /**
  * Reduce the overshooting phenomenon when the wave enters a shallow region
  */
 __global__ void overshootingReduction( const float* depthMap, float* nextDepthMap, const float* heightMap,
                                        const float dx, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0 && i < height && j >= 0 && j < width )
     {
         float hij = map2Dread( depthMap, i,j,width );
         // Exclude the border
         if( i >= 1 && i < height-1 && j >= 1 && j < width-1 )
         {
             // 2.2 section
             const float alpha = 0.3;
             // n(i,j)
             float n = map2Dread( heightMap, i,j,width );
             // n(i-1,j)
             float n2 = map2Dread( heightMap, i-1,j,width );
             // n(i+1,j)
             float n3 = map2Dread( heightMap, i+1,j,width );
             // n(i,j-1)
             float n4 = map2Dread( heightMap, i, j-1, width );
             // n(i,j+1)
             float n5 = map2Dread( heightMap, i, j+1, width );
             float value;
             float nextD = hij;
             float lamda = 2*dx;
             if( n - n2 >lamda && n > n3  )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap,i+1,j,width) ) ) - hij );
                 nextD += value;
             }
             if( n - n3 > lamda && n > n2 )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap,i-1,j,width) ) ) - hij );
                 nextD += value;
             }
             if( n - n4 > lamda && n > n5 )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap, i,j+1,width ) ) ) - hij );
                 nextD += value;
             }
             if( n - n5 > lamda && n > n4 )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap, i, j- 1, width) ) ) - hij );
                 nextD += value;
             }
             map2Dwrite( nextDepthMap, i,j, nextD,width );
         }
         else
         {
             // Just copy
             map2Dwrite( nextDepthMap, i,j, hij,width );
         }
     }
 }

 // Review passed
/**
 * @brief initGrid Initialize our grid
 * @param girdSize The gridSize
 * @param terrainMap The terrainMap from host
 */
void initGridGPU( const int hostGridSize, const float hostdx, const float* hostTerrainMap )
{
    gridSize = hostGridSize;
    uwidth = gridSize + 1;
    uheight = gridSize;
    wwidth = gridSize;
    wheight = gridSize+1;
    mapdx = hostdx;
    mapdxInv = 1.f/mapdx;

    int width = gridSize;
    int height = gridSize;
    // Firstly backup the terrain's heightMap
    error = hipMalloc(&deviceTerrainMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceTerrainMap );
    error = hipMemcpy( deviceTerrainMap, hostTerrainMap, width*height*sizeof(float), hipMemcpyHostToDevice );
    checkCudaError( error );

    // Malloc heightMap
    error = hipMalloc(&deviceHeightMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceHeightMap );

    // Malloc depthMap
    error = hipMalloc(&deviceDepthMap,  width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceDepthMap );

    // Malloc nextDepthMap
    error = hipMalloc(&deviceNextDepthMap,  width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceNextDepthMap );

    // Malloc the normapMap
    error = hipMalloc(&deviceNormalMap,  width*height*sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( deviceNormalMap );

    width = uwidth;
    height = uheight;
    // Malloc velocityUMap
    error = hipMalloc(&deviceVelocityUMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceVelocityUMap );
    // Malloc nextVelocityUMap
    error = hipMalloc(&deviceNextVelocityUMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceNextVelocityUMap );

    width  =wwidth;
    height = wheight;
    // Malloc velocityWMap
    error = hipMalloc(&deviceVelocityWMap, width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceVelocityWMap );
    // Malloc velocityWMap
    error = hipMalloc(&deviceNextVelocityWMap, width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceNextVelocityWMap );

    // initialize the depth map
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    initDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceDepthMap,deviceTerrainMap,gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    // Copy the depth field to initialize the next depth map
    hipMemcpy(deviceNextDepthMap,deviceDepthMap,gridSize*gridSize*sizeof(float), hipMemcpyDeviceToDevice );

    // Initialize the normal map
    initNormalCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceNormalMap,gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    // Initialize velocity U map
    blockPerGridX = (uwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    initFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityUMap, uwidth, uheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    // Copy the velocity U map to initialize next velocity U map
    hipMemcpy(deviceNextVelocityUMap,deviceVelocityUMap,(uwidth)*uheight*sizeof(float), hipMemcpyDeviceToDevice );


    //checkInitializedDeviceField( deviceVelocityUMap, gridSize+1, gridSize );
    //checkInitializedDeviceField( deviceNextVelocityUMap, gridSize+1, gridSize );


    // Initialize velocityW
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    initFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityWMap, wwidth, wheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    // Copy the velocity W map to initialize next velocity W map
    hipMemcpy(deviceNextVelocityWMap,deviceVelocityWMap,(wwidth)*wheight*sizeof(float), hipMemcpyDeviceToDevice );


    //checkInitializedDeviceField( deviceVelocityWMap, wwidth, wheight );
    //checkInitializedDeviceField( deviceNextVelocityWMap, wwidth, wheight );


    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceHeightMap,deviceDepthMap,deviceTerrainMap,
                                                   gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

// Review passed
/**
 * @brief updateFluidGPU Update function interface
 */
void updateFluidGPU( const float dt )
{
    /**
     * Advect the depth
     */
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    advectDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceNextDepthMap, deviceVelocityUMap,deviceVelocityWMap,
                     gridSize,gridSize,dt,mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    /**
     * Copy back the depth
     */
    hipMemcpy( deviceDepthMap, deviceNextDepthMap,
                gridSize*gridSize*sizeof(float), hipMemcpyDeviceToDevice);
    /**
     * Advect the velocity U
     */
    blockPerGridX = (uwidth+blockSizeX-1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY-1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    advectVelUCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityUMap, deviceNextVelocityUMap,
                                                       deviceVelocityWMap, uwidth,uheight, dt,mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Copy back the velocity U
     */
    hipMemcpy( deviceVelocityUMap, deviceNextVelocityUMap,
                uwidth*uheight*sizeof(float), hipMemcpyDeviceToDevice);
    /**
     * Advect the velocity W
     */
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    advectVelWCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityWMap, deviceNextVelocityWMap,
                    deviceVelocityUMap, wwidth, wheight, dt, mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Copy back the velocity W
     */
    hipMemcpy( deviceVelocityWMap, deviceNextVelocityWMap, wwidth*wheight*sizeof(float),hipMemcpyDeviceToDevice );

    /**
     * Update the depth
     */
    blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updateDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap,
                                                        deviceVelocityUMap, deviceVelocityWMap, gridSize,gridSize,dt,mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the height
     */
    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceHeightMap,
                                                         deviceDepthMap, deviceTerrainMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the velocity U
     */
    blockPerGridX = (uwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updateVelUCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityUMap, deviceHeightMap, uwidth, uheight, dt, mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the velocity W
     */
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updateVelWCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityWMap, deviceHeightMap, wwidth, wheight, dt, mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Apply the boundary
     */
    blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    applyBoundaryCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceHeightMap, deviceTerrainMap, gridSize,gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    overshootingReduction<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap,
                                                              deviceNextDepthMap, deviceHeightMap, mapdx, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    /**
     * Copy back the buffer into deviceDepthMap
     */
    hipMemcpy( deviceDepthMap, deviceNextDepthMap, gridSize*gridSize*sizeof(float),hipMemcpyDeviceToDevice );

    /**
     * Apply boundary again
     **/
    applyBoundaryCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceHeightMap, deviceTerrainMap, gridSize,gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * update the height map
     */
    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Compute he normal map
     */
    computeNormalCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceNormalMap, deviceHeightMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

/**
 * @brief addDropGPU Add drop interface
 * @param posX The x coordinate
 * @param posZ The y coordinate
 * @param radius The radius
 * @param h The height added
 */
void addDropGPU(const int posX, const int posZ, const int radius, const float h )
{
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    addDropCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceDepthMap,posX,posZ,radius,h,gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceHeightMap,deviceDepthMap,deviceTerrainMap,
                                                   gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

 /*   computeNormalCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceNormalMap, deviceHeightMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);*/
}

/**
 * @brief destroyCUDAmem Destroy the cuda memory
 */
void destroyGPUmem()
{
    hipFree( deviceNormalMap );
    hipFree( deviceTerrainMap );
    hipFree( deviceHeightMap );
    hipFree( deviceDepthMap );
    hipFree( deviceVelocityUMap );
    hipFree( deviceVelocityWMap );
    hipFree( deviceNextDepthMap );
    hipFree( deviceNextVelocityUMap );
    hipFree( deviceNextVelocityWMap );
    hipDeviceReset();
    hipDeviceReset();
}

/**
 * @brief copyback After each update we need to copy back the map
 * @param host The target host map
 */
void copybackGPU(FieldType type, float* hostMap  )
{
    switch( type )
    {
    case HEIGHT:
    {
        error = hipMemcpy( hostMap, deviceHeightMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case DEPTH:
    {
        error = hipMemcpy( hostMap, deviceDepthMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case NORMAL:
    {
        error = hipMemcpy( hostMap, deviceNormalMap, gridSize*gridSize*sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    default:
    {
        assert(0);
        break;
    }
    }
    checkCudaError( error );
}

/**
 * @brief findSupportGPU Find supported CUDA device counts
 * @return True if device count is not zero
 */
bool findSupportDevice()
{
       int deviceCount = 0;

       hipError_t error_id = hipGetDeviceCount(&deviceCount);

       if (error_id != hipSuccess)
       {
           printf("hipGetDeviceCount returned error code: %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
           printf("> FAILED %s sample finished, exiting...\n" );
           // I turn on the exit, it will never return
           exit(EXIT_FAILURE);
           return false;
       }
       if (deviceCount == 0)
       {
           printf("> There are no device(s) supporting CUDA\n");
           return false;
       }
       else
       {
           printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
           return true;
       }
}
#endif
