#include "hip/hip_runtime.h"
/** fluid_compute.cu
 ** Brief: Deal with all the computation here
 ** Project: large-scale fluids
 ** Date: 04/10/2013
 ** Member: Scott, Hobarts, Yan Li
 **/

#ifndef FLUID_COMPUTE_CU
#define FLUID_COMPUTE_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include "fluid_global.h"
// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define EPS 0.001
extern "C"
{
void initGridGPU( const int hostGridSize, const float hostdx, const float halfdm, const float* hostTerrainMap );
void copybackGPU(FieldType type, float* hostMap  );
void destroyGPUmem();
void addDropGPU(const int posX, const int posZ, const int radius, const float h );
void advectGPU(const float dt);
void updateFluidGPU( const float dt );
bool findSupportDevice();

void initParticlesGPU(const int numParticles);
void updateParticlesGPU( const float dt, const float accX, const float accY, const float accZ );
void intersectParticlesGPU( const float halfDomain, const float mdxInv, const float heightChange );
void inputParticlesGPU(const float *particlePositions, const float *particleVelocities );

void clampFieldsGPU( const float velocityClamp );

void initDampeningFieldsGPU( const int sizeDampeningRegion, const float quadraticA, const float quadraticB, const float quadraticC );
void dampenWavesGPU( const float hRest, const float dt, const float dxInv, const float lambdaUpdate, const float lambdaDecay );
}

__host__ __device__ inline float cudaMax( float a, float b )
{
    return (a>b)?(a):(b);
}

__host__ __device__ inline float cudaMin( float a, float b )
{
    return (a<b)?(a):(b);
}


// The 2D vector structure for GPU computing
 struct vec2
{
    union
    {
        struct {float x,y; };
        float xy[2];
    };
};

 struct vec3
{
    union
    {
        struct {float x,y,z; };
        float xyz[3];
    };
};

const int blockSizeX = 4;
const int blockSizeY = 4;

vec3* devicePaintMap; // Paint map, we copy back this buffer for drawing. It stores the position of vertices
vec3* deviceNormalMap; // Normal map for GPU
float* deviceTerrainMap; // Terrain map for GPU
float* deviceHeightMap; // Height map for GPU
float* deviceDepthMap; // Depth map for GPU
float* deviceVelocityUMap; // VelocityU map for GPU
float* deviceVelocityWMap; // VelocityW map for GPU

float* deviceNextDepthMap; // Temp buffer for storing next depth map
float* deviceNextVelocityUMap; // Temp buffer for storing next velocity U map
float* deviceNextVelocityWMap; // Temp buffer for storing next velocity W map

//particle data structures
vec3* deviceParticlePositionsArray; // particle positions array
vec3* deviceParticleVelocitiesArray; // particle velocities array
int deviceNumParticles; // number of particles

//dampening waves data structures
float* deviceSigmaMap;
float* deviceGammaMap;
float* devicePhiMap;
float* devicePsiMap;
int deviceDampeningRegion; // size of dampening region

/**
 * pitches for the maps above
 */
// Error
hipError_t error;

// The grid size for heightmap, depthmap, terrainmap
int gridSize;
// The width for velocity u
int uwidth;
// The height for veloctiy u
int uheight;
// The width for velocity w
int wwidth;
// The height for veloctiy w
int wheight;

// dx
float mapdx;
// dxInv
float mapdxInv;
// halfDomain
float halfDomain;

// Forward decaration
void initGridGPU( const int hostGirdSize, const float* hostTerrainMap );
void addDropGPU(const int posX, const int posZ, const int radius, const float h );
void copybackGPU(float* hostHeightMap );
void destroyGPU();
void advectGPU(const float dt);
void updateFluidGPU( const float dt );

//particles forward declarations
void initParticlesGPU(const int numParticles);
void updateParticlesGPU( const float dt, const float accX, const float accY, const float accZ );
void intersectParticlesGPU( const float halfDomain, const float mdxInv, const float heightChange );
void inputParticlesGPU( const float *particlePositions, const float *particleVelocities );

//stabilize forward declarations
void clampFieldsGPU( const float velocityClamp );

//dampening waves forward declarations
void initDampeningFieldsGPU( const int sizeDampeningRegion, const float quadraticA, const float quadraticB, const float quadraticC );
void dampenWavesGPU( const float hRest, const float dt, const float dxInv, const float lambdaUpdate, const float lambdaDecay );

void checkInitializedDeviceField( float* device, int width, int height )
{
    float* host = (float*)malloc(width*height*sizeof(float));
    hipMemcpy( host, device, width*height*sizeof(float),hipMemcpyDeviceToHost);

    for( int i = 0; i < width*height; i++ )
    {
        if( host[i] != 0.f )
        {
            assert(0);
        }
    }
    free(host);
}

void  checkCudaError( hipError_t error )
{
    if( error != hipSuccess )
    {
        //cout <<"CUDA error code: "<<hipGetErrorString(error);
        printf( "CUDA error code: %s\n",hipGetErrorString(error) );
    }
}
/**
 * Check if the pointer is null, if it's null, exit the program
 */
template <class T>
void check1DNotNull(T* array )
{
    if( array == NULL )
    {
        printf("Wrong pointer\n");
    }
}

/**
 * Initialize a vec2
 */
__host__ __device__ inline vec2 initVec2( float x, float y )
{
    vec2 result;
    result.x = x;result.y = y;
    return result;
}

/**
 *  Initialize a vec3
 */
__host__ __device__ inline vec3 initVec3( float x, float y, float z )
{
    vec3 result;
    result.x = x;result.y = y; result.z = z;
    return result;
}

// Review passed
/**
* Compute the cross product of two vectors
*/
__host__ __device__ inline vec3 cross( const vec3 v1, const vec3 v2 )
{
    vec3 result = initVec3(v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z, v1.x * v2.y - v1.y * v2.x);
    return result;
}

// Review passed
/**
 * Compute the normalized vector
 */
__host__ __device__ inline vec3 normalize( const vec3 v )
{
    float d = sqrt(v.x*v.x + v.y*v.y + v.z*v.z );
    return initVec3( v.x/d,v.y/d, v.z/d );
}

// Review passed
/**
 * Functions for getting data from 2D array in GPU
 */
__host__ __device__ inline float map2Dread( const float* map, int i, int j, int width )
{
    return map[i*width+ j];
}
// Review passed
/**
 * Functions for writting data from 2D array in GPU
 */
__host__ __device__ inline void map2Dwrite( float* map, int i, int j, float value, int width )
{
    map[i*width + j] = value;
}

// Review passed
/**
 *  Initialize the depth
 */
__global__ void initDepthCUDA( float* depthMap, const float* terrainMap, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;

    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        float depth = cudaMax(0.f,defaultHeight - map2Dread( terrainMap, i,j, width ));
       // float depth = 5;
        map2Dwrite( depthMap,i,j,depth,width );
    }
}

// Review passed
/**
 * Update the height field by plusing the depth and terrain
 */
__global__ void updateHeightCUDA( float*heightMap, const float* depthMap, const float* terrainMap,
                              const int width, const int height )
{
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        float depth = map2Dread( depthMap,i,j,width );
        float terrainHeight = map2Dread( terrainMap, i,j,width );
        float h = depth + terrainHeight;
        map2Dwrite( heightMap, i,j, h ,width);
    }
}

// Review passed
/**
 * Add drop to specified rectangular region
 */
__global__ void addDropCUDA( float* depthMap, const int posX, const int posZ, const int radius,
                             const float h, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    if( i>= cudaMax(posZ-radius,0) && i < cudaMin(posZ+radius+1,height)
            && j >= cudaMax(posX-radius,0)&&j < cudaMin(posX+radius+1,width)
            )
    {
        float newH = cudaMin(map2Dread(depthMap,i,j,width)+h,maxHeight );
        map2Dwrite( depthMap, i,j, newH, width );
    }
}

// Review passed
/**
 * bilinear interpolation
 */
 __host__ __device__ float bilinearIerp( const float* vec, float x, float z, const int width, const int height )
 {
     if( x < 0 )
         x = 0.f;
     if( z < 0 )
         z = 0.f;
     if( x > width - 1 )
         x = width - 1;
     if( z > height - 1 )
         z = height -1;

     const int X = (int)x;
     const int Y = (int)z;
     const float s1 = x - X;
     const float s0 = 1.f - s1;
     const float t1 = z - Y;
     const float t0 = 1.f-t1;
     float e1, e2, e3,e4;
     e1 = e2 = e3 = e4 = 0;
     //e1 = vec[Y*width+X];
     e1 = map2Dread(vec,Y,X,width);
     if( Y+1 <= height- 1 )
     {
      //   e2 = vec[(Y+1)*width + X];
         e2 = map2Dread( vec, Y+1,X,width);
     }
     if( X +1 <= width -1 )
     {
        // e3 = vec[Y*width + X+1];
         e3 = map2Dread( vec, Y, X+1, width );
     }
     if( Y+1 <= height - 1 && X + 1 <= width - 1)
     {
      //   e4 = vec[(Y+1)*width + X+1];
         e4 = map2Dread( vec, Y+1,x+1,width );
     }

     float result = s0*(t0*e1 + t1*e2 )+
             s1*(t0*e3  + t1*e4 );

     return  result;
 }

 // Review passed
 /**
  * Advection: depth
  **/
 __global__ void advectDepthCUDA( const float* depthMap, float* nextDepthMap, const float* velUMap, const float* velWMap,
                              const int width, const int height, const float dt, const float dxInv)
{
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height -1 && j >= 1 && j < width - 1 )
     {
         // Be careful about the width of velU
         float uw = width + 1;
         float u = 0.5*(map2Dread(velUMap,i,j,uw) + map2Dread( velUMap,i,j+1,uw) );
         float w = 0.5*(map2Dread(velWMap,i,j,width) + map2Dread( velWMap,i+1,j,width) );

         float curPosX = (float)j;
         float curPosY = (float)i;
         float prev_x = curPosX - u*dt*dxInv;
         float prev_z = curPosY - w*dt*dxInv;
         map2Dwrite(nextDepthMap, i,j, bilinearIerp( depthMap,prev_x,prev_z,width,height ), width);
     }
 }

 // Review passed
 /**
  * Advection: velocity U
  */
 __global__ void advectVelUCUDA( const float* velUMap, float* nextVelUMap, const float* velWMap,
                            const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height - 1&&j >= 1 && j < width - 1)
     {
         float ww = width - 1;
        float u = map2Dread( velUMap, i,j,width );
        float w = 0.25*(map2Dread(velWMap,i,j, ww ) + map2Dread( velWMap, i,j-1,ww) + map2Dread( velWMap,i+1,j-1,ww) + map2Dread( velWMap, i+1,j,ww ) );
        float curPosX = (float)j;
        float curPosY = (float)i;
        float prev_x = curPosX - u*dt*dxInv;
        float prev_z = curPosY - w*dt*dxInv;
        map2Dwrite(nextVelUMap, i,j, bilinearIerp(  velUMap,prev_x,prev_z,width,height ), width);
     }
 }

 // Review passed
 /**
  *  Advection: velocity W
  */
 __global__ void advectVelWCUDA( const float* velWMap, float* nextVelWMap, const float* velUMap,
                            const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height - 1&&j >= 1 && j < width - 1)
     {
        float uw = width + 1;
        float u = 0.25*(map2Dread( velUMap,i,j,uw) + map2Dread(velUMap,i,j+1,uw) + map2Dread(velUMap,i-1,j+1,uw) + map2Dread(velUMap, i-1,j,uw) );
        float w = map2Dread( velWMap,i,j,width );
        float curPosX = (float)j;
        float curPosY = (float)i;
        float prev_x = curPosX - u*dt*dxInv;
        float prev_z = curPosY - w*dt*dxInv;
        map2Dwrite(nextVelWMap,i,j,bilinearIerp(  velWMap,prev_x,prev_z,width,height ), width);
     }
 }

 // Review passed
 /**
  * Update the depth field
  **/
 __global__ void updateDepthCUDA(float* depthMap, const float* velUMap, const float* velWMap,
                                 const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 1 && i < height-1 && j >= 1 && j < width-1)
     {
            float decay = 1.f;
            float dep = map2Dread(depthMap,i,j,width);
            float dh = -decay*dep*dxInv*( (map2Dread(velUMap,i,j+1,width+1) - map2Dread(velUMap,i,j,width+1))
                                          + (map2Dread(velWMap,i+1,j,width) - map2Dread(velWMap,i,j,width)) );
            float nextDepth = dh*dt+dep;
            if( nextDepth < EPS )
            {
                 map2Dwrite(depthMap, i, j, 0.f, width );
            }
            else
            {
                map2Dwrite(depthMap, i, j, dh*dt+dep, width );
            }
     }
 }

 // Review passed
 /**
  * Update the velocity U field
  */
 __global__ void updateVelUCUDA( float* velUMap, const float* heightMap, const float* depthMap,
                                 const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >=1 && i < height - 1 && j >= 2 && j < width - 2 )
     {
         // The width of heightmap is 1 smaller than the width of velocity U
        float h1 = map2Dread( heightMap, i,j, width - 1);
        float h2 = map2Dread(  heightMap, i,j-1, width-1 );
        float d1 = map2Dread(depthMap,i,j,width-1);
        float d2 = map2Dread(depthMap,i,j-1,width-1);

        // Read the origin value from velUMap
        float vel = map2Dread( velUMap, i,j, width );

        if( d1 < EPS || d2 < EPS )
        {
            float vel1 = map2Dread( velUMap,i,j-1,width);
            float vel2 = map2Dread( velUMap,i,j+1,width);
            float vel3 = map2Dread( velUMap, i,j, width );
            map2Dwrite( velUMap,i,j,0.33*(vel1+vel2+vel3),width);
            return;

        }
        float dv = GRAVITY*dt*dxInv*(h1-h2);

        // Add
        map2Dwrite( velUMap, i,j,vel+dv, width);
     }
     else
     {
         // for the bounday, we set the velocity to zero, using the neuman boundary condition
        map2Dwrite( velUMap, i,j,0.f, width);
     }
 }

 // Review passed
 /**
  * Update the velocity W field
  */
 __global__ void updateVelWCUDA( float* velWMap, const float* heightMap, const float* depthMap,
                                 const int width, const int height, const float dt, const float dxInv )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 2 && i < height - 2 && j >= 1 && j < width - 1 )
     {
         float h1 = map2Dread( heightMap, i,j, width );
         float h2 = map2Dread(  heightMap, i-1,j, width );

         float d1 = map2Dread(depthMap,i,j,width);
         float d2 = map2Dread(depthMap,i-1,j,width);

         float vel = map2Dread( velWMap, i,j, width );

         if( d1 < 0.0001 || d2 < 0.0001 )
         {
             float vel1 = map2Dread( velWMap,i-1,j,width);          
             float vel2 = map2Dread( velWMap,i+1,j,width);
             float vel3 = map2Dread( velWMap, i, j, width );
             map2Dwrite( velWMap,i,j,0.33*(vel1+vel2+vel3),width);
             return;

         }
         float dv = GRAVITY*dt*dxInv*(h1-h2);
         map2Dwrite( velWMap, i,j,vel+dv, width );
     }
     else
     {
         // for the bounday, we set the velocity to zero, using the neuman boundary condition
        map2Dwrite( velWMap, i,j,0.f, width);
     }
 }

 // Review passed
 /**
  * Apply the boundary condition
  **/
 __global__ void applyBoundaryCUDA( float* depthMap, const float* heightMap, const float* terrainMap,
                                    const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
    float value;
    // Fix the boundary
    /*if( i == 0 || i == height-1|| j == 0 ||  j == width-1 )
    {
        map2Dwrite( depthMap, i,j, 0, width );
    }*/
     if( j == 0 && i !=  0 && i != height-1)
     {
         value = max(0.f, map2Dread( heightMap, i,1,width) - map2Dread( terrainMap, i,j,width ) );
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if( j== width - 1&& i !=0 && i != height - 1 )
     {
         value = max( 0.f, map2Dread( heightMap, i, width - 2, width ) - map2Dread( terrainMap, i, j,width) );
        map2Dwrite( depthMap, i, j, value, width );
        return;
     }

     if( i == 0&& j != 0 && j != width -1 )
     {
         value = max(0.f, map2Dread( heightMap, 1, j, width) - map2Dread( terrainMap, i, j,width ));
         map2Dwrite( depthMap, i, j, value, width );
         return;
     }
     else if( i == height - 1&& j != width - 1 && j != 0)
     {
         value = max(0.f, map2Dread( heightMap, height - 2, j, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i, j, value, width );
         return;
     }

     // Deal with the four courner, is there a way to simplify this? This function is too long!
     if( i== 0 && j == 0 )
     {
         value = max(0.f, map2Dread( heightMap, 1, 1, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if( i==0 && j == width - 1 )
     {
         value = max(0.f, map2Dread( heightMap, 1, width-2, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if ( i == height - 1 && j == 0 )
     {
         value = max(0.f, map2Dread( heightMap, height - 2, 1, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
     else if( i == height - 1 && j == width - 1 )
     {
         value = max(0.f, map2Dread( heightMap, height - 2, width-2, width ) - map2Dread(terrainMap, i,j,width ));
         map2Dwrite( depthMap, i,j, value, width );
         return;
     }
 }

 // Review passed
 /**
  * A general function to initialize the field to be all zero( Cannot use memset because we are using float )
  **/
 __global__ void initFieldCUDA( float* deviceMap, int width, int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
         map2Dwrite( deviceMap, i, j, 0.f, width );
     }
 }

/**
 * Initialize the normal field
 **/
 __global__ void initNormalCUDA( vec3* normalMap, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
         const int index = i*width + j;
         normalMap[index].x = 0;
         normalMap[index].y = 1;
         normalMap[index].z = 0;
     }
 }

 /**
  * Initialize the paint field
  */
 __global__ void updatePaintCUDA( vec3* paintMap, const float* heightMap, const float halfdm, const float dx, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
         const int index = i*width + j;
         paintMap[index].x = -halfdm + j*dx;
         paintMap[index].y = heightMap[index];
         paintMap[index].z = -halfdm + i*dx;
     }
 }

 /**
  * compute the normals
  **/
 __global__ void computeNormalCUDA( vec3* normalMap, const float* heightMap, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0&& i < height&&j >= 0 && j < width )
     {
            int numNeighbours = 0;

            vec3 offset[8];
            vec2 coords[8];
            vec2 neighbours[8];

            coords[0] = initVec2(i,     j - 1);
            coords[1] = initVec2(i + 1, j - 1);
            coords[2] = initVec2(i + 1, j);
            coords[3] = initVec2(i + 1, j + 1);
            coords[4] = initVec2(i,     j + 1);
            coords[5] = initVec2(i - 1, j + 1);
            coords[6] = initVec2(i - 1, j);
            coords[7] = initVec2(i - 1, j - 1);
            int m;
            for( m = 0; m < 8; m++ )
            {
                if( coords[m].x < 0 || coords[m].y < 0 || coords[m].x > height- 1 || coords[m].y > width- 1 )
                    continue;
                neighbours[numNeighbours] = coords[m];
                numNeighbours++;
            }

            for( m = 0; m < numNeighbours; m++ )
            {
                offset[m].x = neighbours[m].y - j;
                offset[m].z = neighbours[m].x - i;
                offset[m].y = map2Dread(heightMap,neighbours[m].x,neighbours[m].y,width) - map2Dread( heightMap, i,j,width );
            }

            vec3 sum = initVec3(0.f,0.f,0.f);
            for( m = 0; m < numNeighbours; m++ )
            {
                vec3 tmp;
                if( m+1 == numNeighbours )
                    tmp = cross( offset[m],offset[0]);
                else
                    tmp = cross( offset[m],offset[m+1]);
                sum.x += tmp.x;
                sum.y += tmp.y;
                sum.z += tmp.z;
            }
            vec3 result = normalize( sum );
           const int index = i*width+j;
           normalMap[index].x = result.x; normalMap[index].y = result.y; normalMap[index].z = result.z;
     }
 }

 /**
  * Reduce the overshooting phenomenon when the wave enters a shallow region
  */
 __global__ void overshootingReduction( const float* depthMap, float* nextDepthMap, const float* heightMap,
                                        const float dx, const int width, const int height )
 {
     int i = blockDim.y*blockIdx.y + threadIdx.y;
     int j = blockDim.x*blockIdx.x + threadIdx.x;
     if( i >= 0 && i < height && j >= 0 && j < width )
     {
         float hij = map2Dread( depthMap, i,j,width );
         // Exclude the border
         if( i >= 1 && i < height-1 && j >= 1 && j < width-1 )
         {
             // 2.2 section
             const float alpha = 0.3;
             // n(i,j)
             float n = map2Dread( heightMap, i,j,width );
             // n(i-1,j)
             float n2 = map2Dread( heightMap, i-1,j,width );
             // n(i+1,j)
             float n3 = map2Dread( heightMap, i+1,j,width );
             // n(i,j-1)
             float n4 = map2Dread( heightMap, i, j-1, width );
             // n(i,j+1)
             float n5 = map2Dread( heightMap, i, j+1, width );
             float value;
             float nextD = hij;
             float lamda = 2*dx;
             if( n - n2 >lamda && n > n3  )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap,i+1,j,width) ) ) - hij );
                 nextD += value;
             }
             if( n - n3 > lamda && n > n2 )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap,i-1,j,width) ) ) - hij );
                 nextD += value;
             }
             if( n - n4 > lamda && n > n5 )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap, i,j+1,width ) ) ) - hij );
                 nextD += value;
             }
             if( n - n5 > lamda && n > n4 )
             {
                 value = alpha*( cudaMax( 0.f, 0.5*( hij + map2Dread(depthMap, i, j- 1, width) ) ) - hij );
                 nextD += value;
             }
             map2Dwrite( nextDepthMap, i,j, nextD,width );
         }
         else
         {
             // Just copy
             map2Dwrite( nextDepthMap, i,j, hij,width );
         }
     }
 }

/**
 * Initialize the particle positions field
 **/
__global__ void initParticlePositionsCUDA( vec3* positionsMap, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        positionsMap[i].x = 0;
        positionsMap[i].y = -1;
        positionsMap[i].z = 0;
    }
}

/**
 * Initialize the particle velocities field
 **/
__global__ void initParticleVelocitiesCUDA( vec3* velocitiesMap, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        velocitiesMap[i].x = 0;
        velocitiesMap[i].y = 0;
        velocitiesMap[i].z = 0;
    }
}

/**
 * Update the particle positions and velocities fields
 **/
__global__ void updateParticleValuesCUDA( vec3* positionsMap, vec3* velocitiesMap, float accX, float accY, float accZ, float dt, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        //check if particle is active
        if(positionsMap[i].y >= 0){
            //update position vector
            positionsMap[i].x = positionsMap[i].x +
                    (velocitiesMap[i].x * dt) +
                    (accX * dt * dt);
            positionsMap[i].y = positionsMap[i].y +
                    (velocitiesMap[i].y * dt) +
                    (accY * dt * dt);
            positionsMap[i].z = positionsMap[i].z +
                    (velocitiesMap[i].z * dt) +
                    (accZ * dt * dt);

            //update velocity vector
            velocitiesMap[i].x = velocitiesMap[i].x + (accX * dt);
            velocitiesMap[i].y = velocitiesMap[i].y + (accY * dt);
            velocitiesMap[i].z = velocitiesMap[i].z + (accZ * dt);
        }
    }
}

// TODO: CHECK THIS
// TODO: CONCERNS:
//  1. no concurrency on depth map, might lose contribution
/**
 * Intersect the particles with the height and depth fields
 **/
__global__ void updateParticleValuesCUDA( vec3* positionsMap, float* heightMap, float* depthMap,
                                          const int width, const int height,
                                          const float halfDomain, const float mdxInv, float heightChange, int numParticles )
{
    //index of current vector
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= 0 && i < numParticles ){
        //check if particle is active
        if(positionsMap[i].y >= 0){
            //find grid positions x and z
            float lenX = (positionsMap[i].x + halfDomain) * mdxInv;
            float lenZ = (positionsMap[i].z + halfDomain) * mdxInv;
            int x = (int) cudaMin(width - 1, cudaMax(0.0, round(lenX)));
            int z = (int) cudaMin(height - 1, cudaMax(0.0, round(lenZ)));

            //check if position y < heightMap
            float eta = map2Dread( heightMap, z, x, width );
            if(eta >= positionsMap[i].y){
                float hxz = map2Dread( depthMap, z, x, width );
                map2Dwrite( depthMap, z, x, hxz + heightChange, width );

                positionsMap[i].y = -1;
            }
        }
    }
}

// TODO: CHECK THIS
/**
 *  clamp the depth (min 0)
 */
__global__ void clampDepthCUDA( float* depthMap, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;

    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        float depth = cudaMax(0.f, map2Dread( depthMap, i, j, width ));
        map2Dwrite( depthMap, i, j, depth, width );
    }
}

// TODO: CHECK THIS
/**
 * clamp the velocity (max velocity clamp)
 **/
__global__ void clampFieldCUDA( float* deviceMap, float velocityClamp, int width, int height )
{
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;
    if( i >= 0&& i < height&&j >= 0 && j < width )
    {
        float value = cudaMin(velocityClamp, map2Dread( deviceMap, i, j, width ));
        map2Dwrite( deviceMap, i, j, value, width );
    }
}

//TODO: CHECK THIS
/**
 *  Initialize the sigma and gamma fields
 */
__global__ void initSigmaGammaCUDA( float* sigmaMap, float* gammaMap, const float dampeningRegion,
                                    const float quadraticA, const float quadraticB, const float quadraticC,
                                    const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;
    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        if(i < dampeningRegion || i >= height - dampeningRegion ||
                j < dampeningRegion || j >= width - dampeningRegion){
            //compute horizontal and vertical distances
            float iDistance = 0;
            float jDistance = 0;
            if(i < dampeningRegion){
                iDistance = dampeningRegion - i;
            } else if(i >= height - dampeningRegion){
                iDistance = i - (height - dampeningRegion - 1);
            }

            if(j < dampeningRegion){
                jDistance = dampeningRegion - j;
            } else if(j >= width - dampeningRegion){
                jDistance = j - (width - dampeningRegion - 1);
            }

            iDistance /= (float)dampeningRegion;
            jDistance /= (float)dampeningRegion;

            //distance
            float distance = sqrt((iDistance * iDistance) + (jDistance * jDistance));

            //quadratic function
            float value = (quadraticA * distance * distance) + (quadraticB * distance) + quadraticC;

            //initialize to value
            map2Dwrite( sigmaMap, i, j, value, width );
            map2Dwrite( gammaMap, i, j, value, width );
        } else {
            //initialize to 0
            map2Dwrite( sigmaMap, i, j, 0.0f, width );
            map2Dwrite( gammaMap, i, j, 0.0f, width );
        }
    }
}

/**
 *  Initialize the phi and psi fields
 */
//TODO: CHECK THIS
__global__ void initPhiPsiCUDA( float* phiMap, float* psiMap, const int width, const int height )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;
    if( i >= 0 && i < height && j >= 0 && j < width )
    {
        //initialize to 0
        map2Dwrite( phiMap, i, j, 0.0f, width );
        map2Dwrite( psiMap, i, j, 0.0f, width );
    }
}

/**
 *  Dampen the waves (dampen the depth field), update wave dampening data structures
 */
//TODO: CHECK THIS
__global__ void dampenWavesCUDA( float* depthMap, float* heightMap, float* velUMap, float* velWMap,
                                 float* sigmaMap, float* gammaMap, float* phiMap, float* psiMap,
                                 float dampeningRegion, float hRest, float dt, float dxInv, float lambdaUpdate, float lambdaDecay,
                                 const int width, const int height, const int uwidth, const int uheight, const int wwidth, const int wheight )
{
    int i = blockDim.y*blockIdx.y +threadIdx.y;
    int j = blockDim.x*blockIdx.x +threadIdx.x;

    //TODO: CORRECT ITERATION AREA?
    if( i >= 1 && i < height - 1 && j >= 1 && j < width - 1 )
    {
        if(i < dampeningRegion || i >= height - dampeningRegion ||
                j < dampeningRegion || j >= width - dampeningRegion){
            // current values
            float currH = map2Dread( heightMap, i, j, width );
            float currDepth = map2Dread( depthMap, i, j, width );

            float currVelU = map2Dread( velUMap, i, j, uwidth );
            float currVelUDec = map2Dread( velUMap, i - 1, j, uwidth );
            float currVelW = map2Dread( velWMap, i, j, wwidth );
            float currVelWDec = map2Dread( velWMap, i, j - 1, wwidth );

            float currSigma = map2Dread( sigmaMap, i, j, width );
            float currSigmaInc = map2Dread( sigmaMap, i + 1, j, width );
            float currGamma = map2Dread( gammaMap, i, j, width );
            float currGammaInc = map2Dread( gammaMap, i, j + 1, width );
            float currPhi = map2Dread( phiMap, i, j, width );
            float currPsi = map2Dread( psiMap, i, j, width );

            // Equation 10
            // h(i,j) += ((-sigma(i,j) * (h(i,j) - hRest)) + phi(i,j)) * delta_t
            // Equation 21
            // h(i,j) += ((-gamma(i,j) * (h(i,j) - hRest)) + psi(i,j)) * delta_t
            float eq10 = ((-currSigma * (currH - hRest)) + currPhi) * dt;
            float eq21 = ((-currGamma * (currH - hRest)) + currPsi) * dt;
            map2Dwrite( depthMap, i, j, currDepth + eq10 + eq21, width );

            // Equation 11
            // u(i+0.5,j) += -0.5 * (sigma(i+1,j) + sigma(i,j)) * u(i+0.5,j) * delta_t
            float eq11 = -0.5 * (currSigmaInc + currSigma) * currVelU * dt;
            map2Dwrite( velUMap, i, j, currVelU + eq11, uwidth);

            // Equation 22
            // w(i,j+0.5) += -0.5 * (gamma(i,j+1) + gamma(i,j)) * w(i,j+0.5) * delta_t
            float eq22 = -0.5 * (currGammaInc + currGamma) * currVelW * dt;
            map2Dwrite( velWMap, i, j, currVelW + eq22, wwidth);

            //TODO: reread u and w????

            // Equation 12
            // phi(i,j) += -LAMBDA_UPDATE * sigma(i,j) * ((w(i,j+0.5) - w(i,j-0.5)) / delta_x) * delta_t
            // Equation 13
            // phi(i,j) *= LAMBDA_DECAY
            float eq12 = -lambdaUpdate * currSigma * (currVelW - currVelWDec) * dxInv * dt;
            map2Dwrite( phiMap, i, j, (currPhi + eq12) * lambdaDecay, width );

            // Equation 23
            // psi(i,j) += -LAMBDA_UPDATE * gamma(i,j) * ((u(i+0.5,j) - u(i-0.5,j)) / delta_x) * delta_t
            // Equation 24
            // psi(i,j) *= LAMBDA_DECAY
            float eq23 = -lambdaUpdate * currGamma * (currVelU - currVelUDec) * dxInv * dt;
            map2Dwrite( psiMap, i, j, (currPsi + eq23) * lambdaDecay, width );
        }
    }
}

 // Review passed
/**
 * @brief initGrid Initialize our grid
 * @param girdSize The gridSize
 * @param terrainMap The terrainMap from host
 */
void initGridGPU( const int hostGridSize, const float hostdx, const float halfdm, const float* hostTerrainMap )
{
    gridSize = hostGridSize;
    halfDomain = halfdm;
    uwidth = gridSize + 1;
    uheight = gridSize;
    wwidth = gridSize;
    wheight = gridSize+1;
    mapdx = hostdx;
    mapdxInv = 1.f/mapdx;

    int width = gridSize;
    int height = gridSize;
    // Firstly backup the terrain's heightMap
    error = hipMalloc(&deviceTerrainMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceTerrainMap );
    error = hipMemcpy( deviceTerrainMap, hostTerrainMap, width*height*sizeof(float), hipMemcpyHostToDevice );
    checkCudaError( error );

    // Malloc heightMap
    error = hipMalloc(&deviceHeightMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceHeightMap );

    // Malloc depthMap
    error = hipMalloc(&deviceDepthMap,  width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceDepthMap );

    // Malloc nextDepthMap
    error = hipMalloc(&deviceNextDepthMap,  width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceNextDepthMap );

    // Malloc the normapMap
    error = hipMalloc(&deviceNormalMap,  width*height*sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( deviceNormalMap );

    // Malloc the paintMap
    error = hipMalloc(&devicePaintMap,  width*height*sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( devicePaintMap );

    width = uwidth;
    height = uheight;
    // Malloc velocityUMap
    error = hipMalloc(&deviceVelocityUMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceVelocityUMap );
    // Malloc nextVelocityUMap
    error = hipMalloc(&deviceNextVelocityUMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceNextVelocityUMap );

    width  =wwidth;
    height = wheight;
    // Malloc velocityWMap
    error = hipMalloc(&deviceVelocityWMap, width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceVelocityWMap );
    // Malloc velocityWMap
    error = hipMalloc(&deviceNextVelocityWMap, width*height*sizeof(float));
    checkCudaError( error );
    check1DNotNull( deviceNextVelocityWMap );

    // initialize the depth map
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    initDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceDepthMap,deviceTerrainMap,gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    // Copy the depth field to initialize the next depth map
    hipMemcpy(deviceNextDepthMap,deviceDepthMap,gridSize*gridSize*sizeof(float), hipMemcpyDeviceToDevice );

    // Initialize the normal map
    initNormalCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceNormalMap,gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    // Initialize velocity U map
    blockPerGridX = (uwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    initFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityUMap, uwidth, uheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    // Copy the velocity U map to initialize next velocity U map
    hipMemcpy(deviceNextVelocityUMap,deviceVelocityUMap,(uwidth)*uheight*sizeof(float), hipMemcpyDeviceToDevice );


    //checkInitializedDeviceField( deviceVelocityUMap, gridSize+1, gridSize );
    //checkInitializedDeviceField( deviceNextVelocityUMap, gridSize+1, gridSize );


    // Initialize velocityW
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    initFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityWMap, wwidth, wheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    // Copy the velocity W map to initialize next velocity W map
    hipMemcpy(deviceNextVelocityWMap,deviceVelocityWMap,(wwidth)*wheight*sizeof(float), hipMemcpyDeviceToDevice );


    //checkInitializedDeviceField( deviceVelocityWMap, wwidth, wheight );
    //checkInitializedDeviceField( deviceNextVelocityWMap, wwidth, wheight );


    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceHeightMap,deviceDepthMap,deviceTerrainMap,
                                                   gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    updatePaintCUDA<<<blocksPerGrid,threadsPerBlock>>>( devicePaintMap, deviceHeightMap, halfDomain, mapdx, gridSize,gridSize);
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

// Review passed
/**
 * @brief updateFluidGPU Update function interface
 */
void updateFluidGPU( const float dt )
{
    /**
     * Advect the depth
     */
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    advectDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceNextDepthMap, deviceVelocityUMap,deviceVelocityWMap,
                     gridSize,gridSize,dt,mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    /**
     * Copy back the depth
     */
    hipMemcpy( deviceDepthMap, deviceNextDepthMap,
                gridSize*gridSize*sizeof(float), hipMemcpyDeviceToDevice);
    /**
     * Advect the velocity U
     */
    blockPerGridX = (uwidth+blockSizeX-1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY-1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    advectVelUCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityUMap, deviceNextVelocityUMap,
                                                       deviceVelocityWMap, uwidth,uheight, dt,mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Copy back the velocity U
     */
    hipMemcpy( deviceVelocityUMap, deviceNextVelocityUMap,
                uwidth*uheight*sizeof(float), hipMemcpyDeviceToDevice);
    /**
     * Advect the velocity W
     */
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    advectVelWCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityWMap, deviceNextVelocityWMap,
                    deviceVelocityUMap, wwidth, wheight, dt, mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Copy back the velocity W
     */
    hipMemcpy( deviceVelocityWMap, deviceNextVelocityWMap, wwidth*wheight*sizeof(float),hipMemcpyDeviceToDevice );

    /**
     * Update the depth
     */
    blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updateDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap,
                                                        deviceVelocityUMap, deviceVelocityWMap, gridSize,gridSize,dt,mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the height
     */
    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceHeightMap,
                                                         deviceDepthMap, deviceTerrainMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the velocity U
     */
    blockPerGridX = (uwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updateVelUCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityUMap, deviceHeightMap, deviceDepthMap, uwidth, uheight, dt, mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the velocity W
     */
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    updateVelWCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceVelocityWMap, deviceHeightMap, deviceDepthMap, wwidth, wheight, dt, mapdxInv );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Apply the boundary
     */
/*    blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    applyBoundaryCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceHeightMap, deviceTerrainMap, gridSize,gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
*/
    overshootingReduction<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap,
                                                              deviceNextDepthMap, deviceHeightMap, mapdx, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    /**
     * Copy back the buffer into deviceDepthMap
     */
    hipMemcpy( deviceDepthMap, deviceNextDepthMap, gridSize*gridSize*sizeof(float),hipMemcpyDeviceToDevice );

    /**
     * Apply boundary again
     **/
    applyBoundaryCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceDepthMap, deviceHeightMap, deviceTerrainMap, gridSize,gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * update the height map
     */
    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Compute he normal map
     */
    computeNormalCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceNormalMap, deviceHeightMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    /**
     * Update the paint map
     */
    updatePaintCUDA<<<blocksPerGrid,threadsPerBlock>>>( devicePaintMap, deviceHeightMap, halfDomain, mapdx, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

/**
 * @brief addDropGPU Add drop interface
 * @param posX The x coordinate
 * @param posZ The y coordinate
 * @param radius The radius
 * @param h The height added
 */
void addDropGPU(const int posX, const int posZ, const int radius, const float h )
{
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    addDropCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceDepthMap,posX,posZ,radius,h,gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    updateHeightCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceHeightMap,deviceDepthMap,deviceTerrainMap,
                                                   gridSize,gridSize
                                                   );
    error = hipDeviceSynchronize();
    checkCudaError(error);

 /*   computeNormalCUDA<<<blocksPerGrid,threadsPerBlock>>>( deviceNormalMap, deviceHeightMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);*/
}

/**
 * @brief destroyCUDAmem Destroy the cuda memory
 */
void destroyGPUmem()
{
    hipFree( devicePaintMap );
    hipFree( deviceNormalMap );
    hipFree( deviceTerrainMap );
    hipFree( deviceHeightMap );
    hipFree( deviceDepthMap );
    hipFree( deviceVelocityUMap );
    hipFree( deviceVelocityWMap );
    hipFree( deviceNextDepthMap );
    hipFree( deviceNextVelocityUMap );
    hipFree( deviceNextVelocityWMap );

    hipFree( deviceParticlePositionsArray );
    hipFree( deviceParticleVelocitiesArray );

    hipDeviceReset();
    hipDeviceReset();
}

/**
 * @brief copyback After each update we need to copy back the map
 * @param host The target host map
 */
void copybackGPU(FieldType type, float* hostMap  )
{
    switch( type )
    {
    case HEIGHT:
    {
        error = hipMemcpy( hostMap, deviceHeightMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case DEPTH:
    {
        error = hipMemcpy( hostMap, deviceDepthMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case NORMAL:
    {
        error = hipMemcpy( hostMap, deviceNormalMap, gridSize*gridSize*sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    case PAINT:
    {
         error = hipMemcpy( hostMap, devicePaintMap, gridSize*gridSize*sizeof(vec3), hipMemcpyDeviceToHost);
         break;
    }
    case PARTICLE_POSITIONS:
    {
        error = hipMemcpy( hostMap, deviceParticlePositionsArray, deviceNumParticles * sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    case PARTICLE_VELOCITIES:
    {
        error = hipMemcpy( hostMap, deviceParticleVelocitiesArray, deviceNumParticles * sizeof(vec3), hipMemcpyDeviceToHost);
        break;
    }
    case SIGMA:
    {
        error = hipMemcpy( hostMap, deviceSigmaMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case GAMMA:
    {
        error = hipMemcpy( hostMap, deviceGammaMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case PHI:
    {
        error = hipMemcpy( hostMap, devicePhiMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    case PSI:
    {
        error = hipMemcpy( hostMap, devicePsiMap, gridSize*gridSize*sizeof(float), hipMemcpyDeviceToHost);
        break;
    }
    default:
    {
        assert(0);
        break;
    }
    }
    checkCudaError( error );
}

/**
 * @brief findSupportGPU Find supported CUDA device counts
 * @return True if device count is not zero
 */
bool findSupportDevice()
{
       int deviceCount = 0;

       hipError_t error_id = hipGetDeviceCount(&deviceCount);

       if (error_id != hipSuccess)
       {
           printf("hipGetDeviceCount returned error code: %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
           printf("> FAILED %s sample finished, exiting...\n" );
           // I turn on the exit, it will never return
           exit(EXIT_FAILURE);
           return false;
       }
       if (deviceCount == 0)
       {
           printf("> There are no device(s) supporting CUDA\n");
           return false;
       }
       else
       {
           printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
           return true;
       }
}

void initParticlesGPU(const int numParticles){
    deviceNumParticles = numParticles;

    //malloc the arrays
    error = hipMalloc(&deviceParticlePositionsArray,  deviceNumParticles * sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( deviceParticlePositionsArray );

    error = hipMalloc(&deviceParticleVelocitiesArray,  deviceNumParticles * sizeof(vec3));
    checkCudaError( error );
    check1DNotNull( deviceParticleVelocitiesArray );

    //set up the iterator properties
    int threadsPerBlock = 256;
    int blocksPerGrid = (deviceNumParticles + threadsPerBlock - 1) / threadsPerBlock;

    //initialize positions
    initParticlePositionsCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceParticlePositionsArray,
                                                    deviceNumParticles
                                                    );

    //initialize velocities
    initParticleVelocitiesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceParticleVelocitiesArray,
                                                    deviceNumParticles
                                                    );

    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void updateParticlesGPU( const float dt, const float accX, const float accY, const float accZ ){
    //set up the iterator properties
    int threadsPerBlock = 256;
    int blocksPerGrid = (deviceNumParticles + threadsPerBlock - 1) / threadsPerBlock;

    //update positions and velocities
    updateParticleValuesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceParticlePositionsArray,
                                                    deviceParticleVelocitiesArray,
                                                    accX, accY, accZ, dt,
                                                    deviceNumParticles
                                                    );

    //error check
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

// TODO: CHECK THIS
void intersectParticlesGPU( const float halfDomain, const float mdxInv, const float heightChange ){
    //set up the iterator properties
    int threadsPerBlock = 256;
    int blocksPerGrid = (deviceNumParticles + threadsPerBlock - 1) / threadsPerBlock;

    //intersect particles and velocities
    updateParticleValuesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                                    deviceParticlePositionsArray, deviceHeightMap, deviceDepthMap,
                                                    gridSize, gridSize,
                                                    halfDomain, mdxInv, heightChange, deviceNumParticles
                                                    );

    //set up grid iterator
    dim3 threadsPerBlock2(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid2(blockPerGridX,blockPerGridY);

    //update height field
    updateHeightCUDA<<<blocksPerGrid2, threadsPerBlock2>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap,
                                 gridSize, gridSize );

    //error check
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

void inputParticlesGPU( const float *particlePositions, const float *particleVelocities ){
    //copy over
    error = hipMemcpy( deviceParticlePositionsArray, particlePositions, deviceNumParticles * sizeof(vec3), hipMemcpyHostToDevice );
    checkCudaError( error );

    error = hipMemcpy( deviceParticleVelocitiesArray, particleVelocities, deviceNumParticles * sizeof(vec3), hipMemcpyHostToDevice );
    checkCudaError( error );
}

// TODO: CHECK THIS
void clampFieldsGPU( const float velocityClamp ){
    //TODO: iterate over depth field, make above 0
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);
    //clamp depth, min value is 0
    clampDepthCUDA<<<blocksPerGrid,threadsPerBlock>>>(
                                                   deviceDepthMap, gridSize, gridSize
                                                   );
    //update height field
    updateHeightCUDA<<<blocksPerGrid, threadsPerBlock>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap,
                                 gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);


    //TODO: iterate over velocity U, make below velocity clamp
    blockPerGridX = (uwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (uheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    //clamp velocity u, max value is velocityClamp
    clampFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityUMap, velocityClamp, uwidth, uheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    //TODO: iterate over velocity W, make below velocity clamp
    blockPerGridX = (wwidth + blockSizeX - 1)/(blockSizeX);
    blockPerGridY = (wheight + blockSizeY - 1)/(blockSizeY);
    blocksPerGrid = dim3(blockPerGridX,blockPerGridY);
    //clamp velocity w, max value is velocityClamp
    clampFieldCUDA<<<blocksPerGrid, threadsPerBlock>>>(deviceVelocityWMap, velocityClamp, wwidth, wheight );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

//TODO: CHECK THIS
void initDampeningFieldsGPU( const int sizeDampeningRegion, const float quadraticA, const float quadraticB, const float quadraticC ){
    //set size of dampening region
    deviceDampeningRegion = sizeDampeningRegion;

    //get width and height
    int width = gridSize;
    int height = gridSize;

    //initialize arrays
    //initialize sigma
    error = hipMalloc(&deviceSigmaMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceSigmaMap );

    //initialize gamma
    error = hipMalloc(&deviceGammaMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( deviceGammaMap );

    //initialize phi
    error = hipMalloc(&devicePhiMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( devicePhiMap );

    //initialize psi
    error = hipMalloc(&devicePsiMap, width*height*sizeof(float) );
    checkCudaError( error );
    check1DNotNull( devicePsiMap );

    //fill arrays
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);

    //fill sigma and gamma
    initSigmaGammaCUDA<<<blocksPerGrid, threadsPerBlock>>>( deviceSigmaMap, deviceGammaMap,
                                                    deviceDampeningRegion, quadraticA, quadraticB, quadraticC,
                                                    gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);

    //fill phi and psi
    initPhiPsiCUDA<<<blocksPerGrid, threadsPerBlock>>>( devicePhiMap, devicePsiMap, gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

//TODO: CHECK THIS
void dampenWavesGPU( const float hRest, const float dt, const float dxInv, const float lambdaUpdate, const float lambdaDecay ){
    dim3 threadsPerBlock(blockSizeX,blockSizeY);
    int blockPerGridX = (gridSize + blockSizeX - 1)/(blockSizeX);
    int blockPerGridY = (gridSize + blockSizeY - 1)/(blockSizeY);
    dim3 blocksPerGrid(blockPerGridX,blockPerGridY);

    //dampen waves
    dampenWavesCUDA<<<blocksPerGrid, threadsPerBlock>>>(
                                    deviceDepthMap, deviceHeightMap, deviceVelocityUMap, deviceVelocityWMap,
                                    deviceSigmaMap, deviceGammaMap, devicePhiMap, devicePsiMap,
                                    deviceDampeningRegion, hRest, dt, dxInv, lambdaUpdate, lambdaDecay,
                                    gridSize, gridSize, uwidth, uheight, wwidth, wheight
                                    );
    //update height field
    updateHeightCUDA<<<blocksPerGrid, threadsPerBlock>>>( deviceHeightMap, deviceDepthMap, deviceTerrainMap,
                                 gridSize, gridSize );
    error = hipDeviceSynchronize();
    checkCudaError(error);
}

#endif
