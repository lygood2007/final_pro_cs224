
#ifndef INVERT_CU
#define INVERT_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

extern "C"
void invertImage(unsigned char *bits, int width, int height);

__global__ void invert(unsigned char *bits, int size)
{
    // invert one pixel
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < size)
    {
        for(int i = 0; i < 4; i++)
        {
            bits[4*idx + i] = 0xFF - bits[4*idx + i];
        }
    }
}

void invertImage(unsigned char *bits, int width, int height) {
        unsigned char *device_bits;
        // it's a BGRA, so 4 chars per pixel
        size_t size = 4 * sizeof(unsigned char) * width * height;
        int numpixels = width*height;

        // allocate arrays on device
        hipMalloc((void **) &device_bits, size);

        hipMemcpy(device_bits, bits, size, hipMemcpyHostToDevice);

        // calculation on device
        int blockSize = 32;
        int nBlocks = numpixels/blockSize + (numpixels % blockSize == 0 ? 0 : 1);
        invert <<< nBlocks, blockSize >>> (device_bits, numpixels);
        // retrieve result
        hipMemcpy(bits, device_bits, size, hipMemcpyDeviceToHost);
        // cleanup
        hipFree(device_bits);
}

#endif
